
#include <hip/hip_runtime.h>
#include <hipDNN.h>

#include "../stdafx.h"
#include "../arithmetic.h"

using namespace SN_Base;

/// tensor - input data and output data of each node of the network.

Tensor::Tensor(const snSize& sz) : sz_(sz){

    size_t ssz = sz.size();

    if (ssz > 0){               
        cuAssert(hipMalloc(&dataGPU_, ssz * sizeof(snFloat)));
        cuAssert(hipMemset(dataGPU_, 0, ssz * sizeof(snFloat)));        
    }
}

Tensor::~Tensor(){
    if (dataGPU_)
        cuAssert(hipFree(dataGPU_));

    if (dataCPU_)
        free(dataCPU_);
}

Tensor::Tensor(const Tensor& other){

    setDataGPU(other.getDataGPU(), other.size());
}
      
Tensor& Tensor::operator=(const Tensor& other){

    setDataGPU(other.getDataGPU(), other.size());

    return *this;
}

Tensor& Tensor::operator+=(const Tensor& other){

    ASSERT_MESS(other == *this, "");
       
    summ(sz_, dataGPU_, other.getDataGPU());
   
    return *this;
}

Tensor& Tensor::operator-=(const Tensor& other){

    ASSERT_MESS(other == *this, "");

    difference(sz_, dataGPU_, other.getDataGPU());
    
    return *this;
}

void Tensor::setDataGPU(const snFloat* data, const snSize& nsz){

    size_t nnsz = nsz.size();
    ASSERT_MESS(data && (nnsz > 0), "");

    if (sz_.size() < nnsz){
     
        if (dataGPU_)
            cuAssert(hipFree(dataGPU_));
 
        cuAssert(hipMalloc(&dataGPU_, nnsz * sizeof(snFloat)));
    }

    cuAssert(hipMemcpy(dataGPU_, data, nnsz * sizeof(snFloat), hipMemcpyKind::hipMemcpyDeviceToDevice));
    sz_ = nsz;
}

void Tensor::setDataCPU(const snFloat* data, const snSize& nsz){

    size_t nnsz = nsz.size();
    ASSERT_MESS(data && (nnsz > 0), "");

    if (sz_.size() < nnsz){

        if (dataGPU_)
            cuAssert(hipFree(dataGPU_));

        cuAssert(hipMalloc(&dataGPU_, nnsz * sizeof(snFloat)));
    }

    cuAssert(hipMemcpy(dataGPU_, data, nnsz * sizeof(snFloat), hipMemcpyKind::hipMemcpyHostToDevice));
    sz_ = nsz;
}

snFloat* Tensor::getDataGPU() const{

    return dataGPU_;
}

snFloat* Tensor::getDataCPU() const{
      
    size_t csz = sz_.size();

    dataCPU_ = (snFloat*)realloc(dataCPU_, csz * sizeof(snFloat));

    cuAssert(hipMemcpy(dataCPU_, dataGPU_, csz * sizeof(snFloat), hipMemcpyKind::hipMemcpyDeviceToHost));

    return dataCPU_;
}

void Tensor::resize(const snSize& nsz){

    size_t nnsz = nsz.size(), csz = sz_.size();
    ASSERT_MESS(nnsz > 0, "");

    if (csz < nnsz){

        std::cout << nnsz << std::endl;
        snFloat* mem;
        cuAssert(hipMalloc(&mem, nnsz * sizeof(snFloat)));

        if (dataGPU_){
            if (csz > 0)
               cuAssert(hipMemcpy(mem, dataGPU_, csz * sizeof(snFloat), hipMemcpyKind::hipMemcpyDeviceToDevice));
            cuAssert(hipFree(dataGPU_));
        }
        dataGPU_ = mem;

        cuAssert(hipMemset(dataGPU_ + csz, 0, (nnsz - csz) * sizeof(snFloat)));
    }

    sz_ = nsz;
}