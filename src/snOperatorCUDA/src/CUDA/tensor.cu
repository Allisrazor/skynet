
#include <iostream>

#include <hip/hip_runtime.h>
#include <hipDNN.h>

#include "../stdafx.h"

using namespace SN_Base;

#ifndef cuCHECK
#define cuCHECK(func) if (func != 0){ std::cout << "CUDA error: " << hipGetErrorString(hipGetLastError()) << std::endl; return;}
#endif

/// tensor - input data and output data of each node of the network.

Tensor::Tensor(const snSize& sz) : sz_(sz){

    size_t ssz = sz.size();

    if (ssz > 0){
        cuCHECK(hipMalloc(&data_, ssz * sizeof(snFloat)));
        cuCHECK(hipMemset(data_, 0, ssz * sizeof(snFloat)));
    }
}

Tensor::~Tensor(){
    if (data_)
        cuCHECK(hipFree(data_)); 
}

Tensor::Tensor(const Tensor& other){
    setData(other.getData(), other.size());
}
      
Tensor& Tensor::operator=(const Tensor& other){

    setData(other.getData(), other.size());

    return *this;
}

Tensor& Tensor::operator+=(const Tensor& other){

    assert(other == *this);

    auto od = other.getData();

    //size_t sz = this->size().size();
    //for (size_t i = 0; i < sz; ++i){
    //    data_[i] += od[i];
    //}

    return *this;
}

Tensor& Tensor::operator-=(const Tensor& other){

    assert(other == *this);

    auto od = other.getData();

   /* size_t sz = this->size().size();
    for (size_t i = 0; i < sz; ++i){
        data_[i] -= od[i];
    }*/

    return *this;
}

void Tensor::setData(const snFloat* data, const snSize& nsz){

    size_t nnsz = nsz.size();
    assert(data && (nnsz > 0));

    if (sz_.size() < nnsz){
     
        if (data_)
            cuCHECK(hipFree(data_));
 
        cuCHECK(hipMalloc(&data_, nnsz * sizeof(snFloat)));
    }

    cuCHECK(hipMemcpy(data_, data, nnsz * sizeof(snFloat), hipMemcpyKind::hipMemcpyDeviceToDevice));
    sz_ = nsz;
}

snFloat* Tensor::getData() const{

    return data_;
}

void Tensor::getDataForCPU(snFloat* out, const snSize& osz) const{
    
    assert(sz_ == osz);

    cuCHECK(hipMemcpy(out, data_, sz_.size() * sizeof(snFloat), hipMemcpyDeviceToHost));
}

void Tensor::resize(const snSize& nsz){

    size_t nnsz = nsz.size(), csz = sz_.size();
    assert(nnsz > 0);

    if (csz < nnsz){

        snFloat* mem = nullptr;
        cuCHECK(hipMalloc(&mem, nnsz * sizeof(snFloat)));

        if (data_){
            cuCHECK(hipMemcpy(mem, data_, csz * sizeof(snFloat), hipMemcpyKind::hipMemcpyDeviceToDevice));
            cuCHECK(hipFree(data_));
        }
        data_ = mem;

        cuCHECK(hipMemset(data_ + csz, 0, (nnsz - csz) * sizeof(snFloat)));
    }

    sz_ = nsz;
}

void Tensor::tfree(){
    if (data_)
        cuCHECK(hipFree(data_));

    data_ = nullptr;
    sz_ = snSize(0, 0, 0, 0, 0);
}
