
#include <hip/hip_runtime.h>
#include <hipDNN.h>

#include "../stdafx.h"

using namespace SN_Base;


/// tensor - input data and output data of each node of the network.

Tensor::Tensor(const snSize& sz) : sz_(sz){

    size_t ssz = sz.size();

    if (ssz > 0){               
        cuAssert(hipMalloc(&dataGPU_, ssz * sizeof(snFloat)));
        cuAssert(hipMemset(dataGPU_, 0, ssz * sizeof(snFloat)));        
    }
}

Tensor::~Tensor(){
    if (dataGPU_)
        cuAssert(hipFree(dataGPU_));

    if (dataCPU_)
        free(dataCPU_);
}

Tensor::Tensor(const Tensor& other){

    setDataGPU(other.getDataGPU(), other.size());
}
      
Tensor& Tensor::operator=(const Tensor& other){

    setDataGPU(other.getDataGPU(), other.size());

    return *this;
}

Tensor& Tensor::operator+=(const Tensor& other){

    assert(other == *this);
       
   
    return *this;
}

Tensor& Tensor::operator-=(const Tensor& other){

    assert(other == *this);

    
    return *this;
}

void Tensor::setDataGPU(const snFloat* data, const snSize& nsz){

    size_t nnsz = nsz.size();
    assert(data && (nnsz > 0));

    if (sz_.size() < nnsz){
     
        if (dataGPU_)
            cuAssert(hipFree(dataGPU_));
 
        cuAssert(hipMalloc(&dataGPU_, nnsz * sizeof(snFloat)));
    }

    cuAssert(hipMemcpy(dataGPU_, data, nnsz * sizeof(snFloat), hipMemcpyKind::hipMemcpyDeviceToDevice));
    sz_ = nsz;
}

void Tensor::setDataCPU(const snFloat* data, const snSize& nsz){

    size_t nnsz = nsz.size();
    assert(data && (nnsz > 0));

    if (sz_.size() < nnsz){

        if (dataGPU_)
            cuAssert(hipFree(dataGPU_));

        cuAssert(hipMalloc(&dataGPU_, nnsz * sizeof(snFloat)));
    }

    cuAssert(hipMemcpy(dataGPU_, data, nnsz * sizeof(snFloat), hipMemcpyKind::hipMemcpyHostToDevice));
    sz_ = nsz;
}

snFloat* Tensor::getDataGPU() const{

    return dataGPU_;
}

snFloat* Tensor::getDataCPU() const{
      
    size_t csz = sz_.size();

    dataCPU_ = (snFloat*)realloc(dataCPU_, csz * sizeof(snFloat));

    cuAssert(hipMemcpy(dataCPU_, dataGPU_, csz * sizeof(snFloat), hipMemcpyKind::hipMemcpyDeviceToHost));

    return dataCPU_;
}

void Tensor::resize(const snSize& nsz){

    size_t nnsz = nsz.size(), csz = sz_.size();
    assert(nnsz > 0);

    if (csz < nnsz){

        snFloat* mem = nullptr;
        cuAssert(hipMalloc(&mem, nnsz * sizeof(snFloat)));

        if (dataGPU_){
            cuAssert(hipMemcpy(mem, dataGPU_, csz * sizeof(snFloat), hipMemcpyKind::hipMemcpyDeviceToDevice));
            cuAssert(hipFree(dataGPU_));
        }
        dataGPU_ = mem;

        cuAssert(hipMemset(dataGPU_ + csz, 0, (nnsz - csz) * sizeof(snFloat)));
    }

    sz_ = nsz;
}