
#include <hip/hip_runtime.h>
#include <hipDNN.h>

#include "../stdafx.h"
#include "../structurs.h"

using namespace SN_Base;


void batchNormInit(SN_Base::snFloat* inout, const SN_Base::snSize& iosz, void** pGpuPrm){
        
    bool isFirst = false;

    gpuParams* gpuPrm = (gpuParams*)*pGpuPrm;

    if (!gpuPrm){

        gpuPrm = new gpuParams();
        memset(gpuPrm, 0, sizeof(gpuParams));
        *pGpuPrm = gpuPrm;

        hipdnnHandle_t cudnn = nullptr;
        cuAssert(hipdnnCreate(&cudnn));
        gpuPrm->cudnn = cudnn;      

        isFirst = true;
    }

    if (gpuPrm->inszMem != iosz){
             
        hipdnnActivationMode_t actMode;

        switch (atype){
            case activeType::sigmoid:   actMode = hipdnnActivationMode_t::HIPDNN_ACTIVATION_SIGMOID; break;
            case activeType::relu:      actMode = hipdnnActivationMode_t::HIPDNN_ACTIVATION_RELU; break;
            case activeType::leakyRelu: actMode = hipdnnActivationMode_t::HIPDNN_ACTIVATION_CLIPPED_RELU; break;
            case activeType::elu:       actMode = hipdnnActivationMode_t::HIPDNN_ACTIVATION_ELU; break;
            default:                    actMode = hipdnnActivationMode_t::HIPDNN_ACTIVATION_RELU; break;
        }
        
        // activ_desc
        hipdnnActivationDescriptor_t activ_desc = nullptr;
        cuAssert(hipdnnCreateActivationDescriptor(&activ_desc));
        cuAssert(hipdnnSetActivationDescriptor(activ_desc, actMode, hipdnnNanPropagation_t::HIPDNN_NOT_PROPAGATE_NAN, 0.0));
        if (!isFirst)
            cuAssert(hipdnnDestroyActivationDescriptor(gpuPrm->activ_desc));
        gpuPrm->activ_desc = activ_desc;

        // input
        hipdnnTensorDescriptor_t x_desc = nullptr;
        cuAssert(hipdnnCreateTensorDescriptor(&x_desc));
        cuAssert(hipdnnSetTensor4dDescriptor(x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, int(iosz.n), int(iosz.d), int(iosz.h), int(iosz.w)));
        if (!isFirst)
            cuAssert(hipdnnDestroyTensorDescriptor((hipdnnTensorDescriptor_t)gpuPrm->x_desc));
        gpuPrm->x_desc = x_desc;

        // dinput
        hipdnnTensorDescriptor_t dx_desc = nullptr;
        cuAssert(hipdnnCreateTensorDescriptor(&dx_desc));
        cuAssert(hipdnnSetTensor4dDescriptor(dx_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, int(iosz.n), int(iosz.d), int(iosz.h), int(iosz.w)));
        if (!isFirst)
            cuAssert(hipdnnDestroyTensorDescriptor((hipdnnTensorDescriptor_t)gpuPrm->dx_desc));
        gpuPrm->dx_desc = dx_desc;

        // output
        hipdnnTensorDescriptor_t y_desc;
        cuAssert(hipdnnCreateTensorDescriptor(&y_desc));
        cuAssert(hipdnnSetTensor4dDescriptor(y_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, int(iosz.n), int(iosz.d), int(iosz.h), int(iosz.w)));
        if (!isFirst)
            cuAssert(hipdnnDestroyTensorDescriptor((hipdnnTensorDescriptor_t)gpuPrm->y_desc));
        gpuPrm->y_desc = y_desc;

        // doutput
        hipdnnTensorDescriptor_t dy_desc;
        cuAssert(hipdnnCreateTensorDescriptor(&dy_desc));
        cuAssert(hipdnnSetTensor4dDescriptor(dy_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, int(iosz.n), int(iosz.d), int(iosz.h), int(iosz.w)));
        if (!isFirst)
            cuAssert(hipdnnDestroyTensorDescriptor((hipdnnTensorDescriptor_t)gpuPrm->dy_desc));
        gpuPrm->dy_desc = dy_desc;
       
        gpuPrm->inszMem = iosz;
    }
}

void batchNormFree(uint32_t deviceId, void* pGpuPrm){

    hipSetDevice(deviceId);

    gpuParams* gpuPrm = (gpuParams*)pGpuPrm;

    if (!gpuPrm) return;

    cuAssert(hipdnnDestroy(gpuPrm->cudnn));
    cuAssert(hipdnnDestroyActivationDescriptor(gpuPrm->activ_desc));
    cuAssert(hipdnnDestroyTensorDescriptor(gpuPrm->x_desc));
    cuAssert(hipdnnDestroyTensorDescriptor(gpuPrm->y_desc));
}

void batchNormForward(activeType atype, SN_Base::snFloat* inout, const SN_Base::snSize& iosz, uint32_t deviceId, void** pGpuPrm){
       
    hipSetDevice(deviceId);

    activationInit(atype, inout, iosz, pGpuPrm);

    gpuParams* gpuPrm = (gpuParams*)*pGpuPrm;
      
    cuAssert(hipMemcpy(gpuPrm->in_mem, inout, iosz.size() * sizeof(snFloat), hipMemcpyKind::hipMemcpyDeviceToDevice));

    float alpha = 1.0, beta = 0.0;

    cuAssert(hipdnnActivationForward(gpuPrm->cudnn,
        gpuPrm->activ_desc,
        &alpha,
        gpuPrm->x_desc,
        gpuPrm->in_mem,
        &beta,
        gpuPrm->y_desc,
        inout));        
}

void batchNormBackward(SN_Base::snFloat* inprev, SN_Base::snFloat* outprev, SN_Base::snFloat* inout, const SN_Base::snSize& iosz, uint32_t deviceId, void* pGpuPrm){

    hipSetDevice(deviceId);
    
    gpuParams* gpuPrm = (gpuParams*)pGpuPrm;

    cuAssert(hipMemcpy(gpuPrm->in_mem, inout, iosz.size() * sizeof(snFloat), hipMemcpyKind::hipMemcpyDeviceToDevice));

    float alpha = 1.0, beta = 0.0;

    cuAssert(hipdnnActivationBackward(gpuPrm->cudnn,
                                     gpuPrm->activ_desc,
                                     &alpha,
                                     gpuPrm->y_desc,
                                     outprev,
                                     gpuPrm->dy_desc,
                                     gpuPrm->in_mem,
                                     gpuPrm->x_desc,
                                     inprev,
                                     &beta,
                                     gpuPrm->dx_desc,
                                     inout));
}