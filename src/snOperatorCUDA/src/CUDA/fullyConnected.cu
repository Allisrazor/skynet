#include "hip/hip_runtime.h"
//
// SkyNet Project
// Copyright (C) 2018 by Contributors <https://github.com/Tyill/skynet>
//
// This code is licensed under the MIT License.
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files(the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and / or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions :
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
// THE SOFTWARE.
//

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "../stdafx.h"
#include "snOperatorCUDA/src/Operator/fullyConnected.h"

using namespace std;
using namespace SN_Base;
        
struct gpuParams{

    hipblasHandle_t cuBLAS = 0;
     
};

void FullyConnected::iniParamCUDA(bool isLern, const snSize& insz, size_t kernel, void** pGpuPrm){
    
    size_t ida = insz.w * insz.h * insz.d, bsz = insz.n;

    gpuParams* gpuPrm = (gpuParams*)*pGpuPrm;
    if (!gpuPrm){
    
        hipDeviceProp_t cu_deviceProps;
        hipGetDeviceProperties(&cu_deviceProps, 0);
        if (cu_deviceProps.major < 3){
            ERROR_MESS("%s requires SM >= 3.0");
            return;
        }
        gpuPrm = new gpuParams();
        memset(gpuPrm, 0, sizeof(gpuParams));
        *pGpuPrm = gpuPrm;

        hipblasHandle_t cuHandle = nullptr;
        cuCHECK(hipblasCreate(&cuHandle));

        gpuPrm->cuBLAS = cuHandle;
    }
}
         
void FullyConnected::freeParamCUDA(void* gpuPrms){
    
    gpuParams* gpuPrm = (gpuParams*)gpuPrms;

    if (!gpuPrm) return;
       
    hipblasDestroy(gpuPrm->cuBLAS);
}

__global__ void cuFwdBias(size_t kernel, snSize insz, snFloat* weight, snFloat* output){
       
    weight += insz.w * insz.h * insz.d * kernel;
   
    snFloat* out = output + kernel * blockIdx.x;
    unsigned int k = threadIdx.x;
    while (k < kernel){

        out[k] += weight[k];

        k += blockDim.x;
    }   
}

void FullyConnected::forwardCUDA(size_t kernel, const snSize& insz, const snFloat* input, const snFloat* weight, snFloat* output, void* gpuPrms){
    
    gpuParams* gpuPrm = (gpuParams*)gpuPrms;
    int ida = int(insz.w * insz.h * insz.d), bsz = int(insz.n), krn = int(kernel);
   
  //  cuCHECK(hipblasSetMatrix(bsz, ida, sizeof(snFloat), input, bsz, gpuPrm->d_in, bsz));
  
   // cuCHECK(hipMemcpy(gpuPrm->d_w, weight, (ida + 1) * krn * sizeof(snFloat), hipMemcpyHostToDevice));

    // Out = α * W * In + βC
    // In - data input matrix - values from the previous layer
    // W - weights matrix
    // Out - output matrix
    float alpha = 1.0f, beta = 0.0f;
    cuCHECK(hipblasSgemm(gpuPrm->cuBLAS,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        krn,                           // W, cols
        bsz,                           // In, rows
        ida,                           // In, cols, В М - rows            
        &alpha,                        // α
        weight,                        // W
        krn,                           // W, step to next W (W21 - W11)
        input,                         // In
        ida,                           // In, step to next X (X21 - X11)  
        &beta,                         // β
        output,                        // Out
        krn));                         // Out, step to next Y (Y21 - Y11) 
    
    // +bias
//    cuFwdBias <<< int(insz.n), 128 >>> (kernel, insz, gpuPrm->d_w, gpuPrm->d_out);
    
}

__global__ void cuBwdBias(size_t kernel, snSize insz, snFloat* gradIn, snFloat* dWOut){
    
    dWOut += insz.w * insz.h * insz.d * kernel;
    unsigned int k = threadIdx.x;
    while (k < kernel){
   
        snFloat* grin = gradIn + k, b = 0;
        for (size_t j = 0; j < insz.n; ++j)
            b += grin[kernel * j];

        dWOut[k] = b / insz.n;
        k += blockDim.x;
    }
}

void FullyConnected::backwardCUDA_GW(size_t kernel, const snFloat* weight,
    const snSize& insz, const snFloat* input, const snFloat* gradIn, snFloat* gradOut, snFloat* dWOut, void* gpuPrms){
  
    gpuParams* gpuPrm = (gpuParams*)gpuPrms;
    int ida = int(insz.w * insz.h * insz.d), bsz = int(insz.n), krn = int(kernel);
   /*
    cuCHECK(hipblasSetMatrix(bsz, ida, sizeof(snFloat), input, bsz, gpuPrm->d_in, bsz));
  
    cuCHECK(hipblasSetMatrix(bsz, krn, sizeof(snFloat), gradIn, bsz, d_grin, bsz));
*/

    // Weight gradient
    // dW = αIn^T * GrIn + βdW
    // In - data input matrix from previous layer
    // GrIn - gradient matrix from the next layer
    float alpha = 1.0F / insz.n, beta = 0.0f;
    cuCHECK(hipblasSgemm(gpuPrm->cuBLAS,
        HIPBLAS_OP_N,
        HIPBLAS_OP_T,
        krn,                     // GrIn, cols
        ida,                     // In, cols (+1 - X0)      
        bsz,                     // In, rows
        &alpha,                  // α                
        gradIn,                  // GrIn
        krn,                     // GrIn, step to next 
        input,                   // In
        ida,                     // In, step to next  X (X21 - X11)  
        &beta,                   // β               
        dWOut,                   // dW            
        krn));                   // dW, step to next 
 
    // bias
//    cuBwdBias <<< 1, 128 >>> (kernel, insz, d_grin, gpuPrm->d_dw);
     
//    cuCHECK(hipblasSetMatrix(ida, krn, sizeof(snFloat), weight, ida, gpuPrm->d_w, ida));

    //// Gradient for previous layer
    //// GrOut = αGrIn * W^T + βGrOut
    //// GrIn - gradient matrix from the next layer
    //// W - weight
    alpha = 1.F;
    cuCHECK(hipblasSgemm(gpuPrm->cuBLAS,
        HIPBLAS_OP_T,
        HIPBLAS_OP_N,
        ida,                     // W, cols     
        bsz,                     // W, rows
        krn,                     // GrIn, cols
        &alpha,                  // α                               
        weight,                  // W
        krn,                     // W, step to next 
        gradIn,                  // GrIn
        krn,                     // GrIn, step to next 
        &beta,                   // β               
        gradOut,                 // GrOut                                  
        ida));                   // GrOut, step to next 
    
}

void FullyConnected::backwardCUDA_G(size_t kernel, const snFloat* weight, const snSize& insz, const snFloat* gradIn, snFloat* gradOut, void* gpuPrms){
        
    gpuParams* gpuPrm = (gpuParams*)gpuPrms;
    int ida = int(insz.w * insz.h * insz.d), bsz = int(insz.n), krn = int(kernel);
        
    /*cuCHECK(hipblasSetMatrix(bsz, krn, sizeof(snFloat), gradIn, bsz, d_grin, bsz));

    cuCHECK(hipblasSetMatrix(ida, krn, sizeof(snFloat), weight, ida, gpuPrm->d_w, ida));
*/

    //// Gradient for previous layer
    //// GrOut = αGrIn * W^T + βGrOut
    //// GrIn - gradient matrix from the next layer
    //// W - weight
    float alpha = 1.0F, beta = 0.0f;
    cuCHECK(hipblasSgemm(gpuPrm->cuBLAS,
        HIPBLAS_OP_T,
        HIPBLAS_OP_N,
        ida,                     // W, cols
        bsz,                     // W, rows
        krn,                     // GrIn, cols
        &alpha,                  // α                               
        weight,                  // W
        krn,                     // W, step to next 
        gradIn,                  // GrIn
        krn,                     // GrIn, step to next 
        &beta,                   // β         
        gradOut,                 // GrOut                          
        ida));                   // GrOut, step to next 
        
}