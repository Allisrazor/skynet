#include "hip/hip_runtime.h"
//
// SkyNet Project
// Copyright (C) 2018 by Contributors <https://github.com/Tyill/skynet>
//
// This code is licensed under the MIT License.
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files(the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and / or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions :
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
// THE SOFTWARE.
//
#include "../stdafx.h"
#include "snOperatorCUDA/src/Operator/lossFunction.h"

using namespace std;
using namespace SN_Base;


__global__ void softMaxACrossEntropy(snSize iosz, snFloat* inout){
      
    size_t inStepByD = iosz.w * iosz.h,     // step out by input
           inStepByN = inStepByD * iosz.d;  // step out by batch       

    // gridDim.x - number of out layers
    // gridDim.y - batch size

    inout += blockIdx.x * inStepByN;
           
    __shared__ int tmax;
    __shared__ snFloat tsumm;

    tmax = 0;
    tsumm = 0;

    __syncthreads();

    unsigned int i = threadIdx.x;

    while (i < inStepByN){

        atomicMax(&tmax, int(inout[i]));
       
        i += blockDim.x;
    }

    __syncthreads();
    
    while (i < inStepByN){
       
        inout[i] = (inout[i] - tmax > -20) ? exp(inout[i] - tmax) : 0.1E-8F;

        atomicAdd(&tsumm, inout[i]);
             
        i += blockDim.x;
    }

    __syncthreads();

    while (i < inStepByN){

        inout[i] /= tsumm;

        i += blockDim.x;
    }   
}

void lossForward(const snSize& insz, snFloat* inout, lossType loss){

    dim3 dimBlock(256);
    dim3 dimGrid(int(insz.n));

    switch (loss){
        case lossType::softMaxACrossEntropy:
            softMaxACrossEntropy <<<dimGrid, dimBlock >>>(insz, inout);
            break;
    }
}

void lossBackward(const Tensor& inTns, snFloat* out, snFloat* targ, snFloat* grad, lossType loss){

    
   
}
