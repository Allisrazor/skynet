#include "hip/hip_runtime.h"
//
// SkyNet Project
// Copyright (C) 2018 by Contributors <https://github.com/Tyill/skynet>
//
// This code is licensed under the MIT License.
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files(the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and / or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions :
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
// THE SOFTWARE.
//
#include "../stdafx.h"
#include "snOperatorCUDA/src/Operator/lossFunction.h"

using namespace std;
using namespace SN_Base;


__global__ void softMaxACrossEntropyFwd(snSize iosz, snFloat* inout){
      
    size_t inStepByD = iosz.w * iosz.h,     // step out by input
           inStepByN = inStepByD * iosz.d;  // step out by batch       

    // gridDim.x - number of out layers
 
    inout += blockIdx.x * inStepByN;
           
    __shared__ int tmax;
    __shared__ snFloat tsumm;
    extern __shared__ snFloat sdata[];

    tmax = 0;
    tsumm = 0;
    sdata[threadIdx.x] = 0;

    __syncthreads();

    unsigned int i = threadIdx.x;
    while (i < inStepByN){

        if (inout[i] > sdata[i])
           sdata[i] = inout[i];       

        __syncthreads();

        unsigned int s = ((inStepByN - i) >= blockDim.x) ? blockDim.x / 2 : (inStepByN - i) / 2;
        for (; s > 0; s >>= 1) {
            if ((i < s) && (sdata[i] < sdata[i + s])) {
                sdata[i] = sdata[i + s];
            }
            __syncthreads();
        }
       
        i += blockDim.x;
    }
    tmax = sdata[0];
        
    i = threadIdx.x;
    while (i < inStepByN){
       
        sdata[i] = ((inout[i] - tmax / 100.F) > -20) ? exp(inout[i] - tmax / 100.F) : 0.1E-8F;

        __syncthreads();

        unsigned int s = ((inStepByN - i) >= blockDim.x) ? blockDim.x / 2 : (inStepByN - i) / 2;
        for (; s > 0; s >>= 1) {
            if (i < s) {
                sdata[i] += sdata[i + s];
            }
            __syncthreads();
        }
                     
        i += blockDim.x;
    }
    tsumm = sdata[0];

    i = threadIdx.x;
    while (i < inStepByN){

        inout[i] /= tsumm;

        i += blockDim.x;
    }   
}

__global__ void softMaxACrossEntropyBwd(snSize iosz, snFloat* out, snFloat* targ, snFloat* grad){

    size_t inStepByD = iosz.w * iosz.h,     // step out by input
           inStepByN = inStepByD * iosz.d;  // step out by batch       

    // gridDim.x - number of out layers
    // gridDim.y - batch size  
    
    grad += blockIdx.x * inStepByD + blockIdx.y * inStepByN;
    out += blockIdx.x * inStepByD + blockIdx.y * inStepByN;
    targ += blockIdx.x * inStepByD + blockIdx.y * inStepByN;

    unsigned int i = threadIdx.x;

    while (i < inStepByD){

        grad[i] = out[i] - targ[i];

        i += blockDim.x;
    } 
}

__global__ void binaryCrossEntropyBwd(snSize iosz, snFloat* out, snFloat* targ, snFloat* grad){

    size_t inStepByD = iosz.w * iosz.h,     // step out by input
           inStepByN = inStepByD * iosz.d;  // step out by batch       

    // gridDim.x - number of out layers
    // gridDim.y - batch size  

    grad += blockIdx.x * inStepByD + blockIdx.y * inStepByN;
    out += blockIdx.x * inStepByD + blockIdx.y * inStepByN;
    targ += blockIdx.x * inStepByD + blockIdx.y * inStepByN;

    unsigned int i = threadIdx.x;

    while (i < inStepByD){
        
        grad[i] = (out[i] - targ[i]) / (out[i] * (1.F - out[i]));

        i += blockDim.x;
    }
}

__global__ void regressionMSEBwd(snSize iosz, snFloat* out, snFloat* targ, snFloat* grad){

    size_t inStepByD = iosz.w * iosz.h,     // step out by input
        inStepByN = inStepByD * iosz.d;  // step out by batch       

    // gridDim.x - number of out layers
    // gridDim.y - batch size  

    grad += blockIdx.x * inStepByD + blockIdx.y * inStepByN;
    out += blockIdx.x * inStepByD + blockIdx.y * inStepByN;
    targ += blockIdx.x * inStepByD + blockIdx.y * inStepByN;

    unsigned int i = threadIdx.x;

    while (i < inStepByD){
        
        grad[i] = 2 * (out[i] - targ[i]) / inStepByN;

        i += blockDim.x;
    }
}


void lossForward(const snSize& sz, snFloat* inout, lossType loss){

    dim3 dimBlock(256);
    dim3 dimGrid(int(sz.n));

    switch (loss){
        case lossType::softMaxACrossEntropy:
            softMaxACrossEntropyFwd <<<dimGrid, dimBlock, 256 >>>(sz, inout);
            break;

        case lossType::binaryCrossEntropy:
            break;

        case lossType::regressionMSE: 
            break;
    }
}

void lossBackward(const snSize& sz, snFloat* out, snFloat* targ, snFloat* grad, lossType loss){

    dim3 dimBlock(128);
    dim3 dimGrid(int(sz.d), int(sz.n));

    switch (loss){
      case lossType::softMaxACrossEntropy:
          
          softMaxACrossEntropyBwd << <dimGrid, dimBlock >> >(sz, out, targ, grad); 
          break;    
      
      case lossType::binaryCrossEntropy:
      
          binaryCrossEntropyBwd << <dimGrid, dimBlock >> >(sz, out, targ, grad);
          break;
                                           
      case lossType::regressionMSE: // Mean Square Error
      
          regressionMSEBwd << <dimGrid, dimBlock >> >(sz, out, targ, grad);
          break;
    }   
}
