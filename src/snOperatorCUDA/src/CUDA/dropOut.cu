
#include <hip/hip_runtime.h>
#include <hipDNN.h>

#include "../stdafx.h"

using namespace SN_Base;

struct gpuParams{

    hipdnnHandle_t cudnn = 0;
    hipdnnDropoutDescriptor_t drop_desc = 0;
    hipdnnTensorDescriptor_t in_desc = 0;
    hipdnnTensorDescriptor_t out_desc = 0;
   
    size_t wsSz = 0;
    size_t state_sizes = 0;
    SN_Base::snSize inszMem = 0;

    snFloat* in_mem = nullptr;

    void* state_memory = 0;   
    void* d_ws = 0;
};

void dropOutInit(void** pGpuPrm, SN_Base::snFloat dropOut, SN_Base::snFloat* inout, const SN_Base::snSize& outsz){
        
    bool isFirst = false;

    gpuParams* gpuPrm = (gpuParams*)*pGpuPrm;

    if (!gpuPrm){

        gpuPrm = new gpuParams();
        memset(gpuPrm, 0, sizeof(gpuParams));
        *pGpuPrm = gpuPrm;

        hipdnnHandle_t cudnn = nullptr;
        cuAssert(hipdnnCreate(&cudnn));
        gpuPrm->cudnn = cudnn;

        cuAssert(hipdnnDropoutGetStatesSize(gpuPrm->cudnn, &gpuPrm->state_sizes));
        cuAssert(hipMalloc(&gpuPrm->state_memory, gpuPrm->state_sizes));

        isFirst = true;
    }

    if (gpuPrm->inszMem != outsz){

        // drop_desc
        hipdnnDropoutDescriptor_t drop_desc = nullptr;
        cuAssert(hipdnnCreateDropoutDescriptor(&drop_desc));
        cuAssert(hipdnnSetDropoutDescriptor(drop_desc, gpuPrm->cudnn, dropOut, gpuPrm->state_memory, gpuPrm->state_sizes, 1234ULL));
        if (!isFirst)
            cuAssert(hipdnnDestroyDropoutDescriptor(gpuPrm->drop_desc));
        gpuPrm->drop_desc = drop_desc;

        // input
        hipdnnTensorDescriptor_t in_desc = nullptr;
        cuAssert(hipdnnCreateTensorDescriptor(&in_desc));
        cuAssert(hipdnnSetTensor4dDescriptor(in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, int(outsz.n), int(outsz.d), int(outsz.h), int(outsz.w)));
        if (!isFirst)
            cuAssert(hipdnnDestroyTensorDescriptor((hipdnnTensorDescriptor_t)gpuPrm->in_desc));
        gpuPrm->in_desc = in_desc;

        // output
        hipdnnTensorDescriptor_t out_desc;
        cuAssert(hipdnnCreateTensorDescriptor(&out_desc));
        cuAssert(hipdnnSetTensor4dDescriptor(out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, int(outsz.n), int(outsz.d), int(outsz.h), int(outsz.w)));
        if (!isFirst)
            cuAssert(hipdnnDestroyTensorDescriptor((hipdnnTensorDescriptor_t)gpuPrm->out_desc));
        gpuPrm->out_desc = out_desc;

        // ws
        cuAssert(cudnnDropoutGetReserveSpaceSize(in_desc, &gpuPrm->wsSz));
        cuAssert(hipMalloc(&gpuPrm->d_ws, gpuPrm->wsSz));

        // in mem
        if (!isFirst)
            cuAssert(hipFree(gpuPrm->in_mem));
        cuAssert(hipMalloc(&gpuPrm->in_mem, outsz.size() * sizeof(snFloat)));

        gpuPrm->inszMem = outsz;
    }
}

void dropOutFree(uint32_t deviceId, void* pGpuPrm){

    hipSetDevice(deviceId);

    gpuParams* gpuPrm = (gpuParams*)pGpuPrm;

    if (!gpuPrm) return;

    cuAssert(hipdnnDestroy(gpuPrm->cudnn));
    cuAssert(hipdnnDestroyDropoutDescriptor(gpuPrm->drop_desc));
    cuAssert(hipdnnDestroyTensorDescriptor(gpuPrm->in_desc));
    cuAssert(hipdnnDestroyTensorDescriptor(gpuPrm->out_desc));

    cuAssert(hipFree(gpuPrm->d_ws));
    cuAssert(hipFree(gpuPrm->state_memory));
}

void dropOutForward(SN_Base::snFloat dropOut, SN_Base::snFloat* inout, const SN_Base::snSize& outsz, uint32_t deviceId, void** pGpuPrm){
       
    hipSetDevice(deviceId);

    dropOutInit(pGpuPrm, dropOut, inout, outsz);

    gpuParams* gpuPrm = (gpuParams*)*pGpuPrm;
  
    cuAssert(hipMemcpy(gpuPrm->in_mem, inout, outsz.size() * sizeof(snFloat), hipMemcpyKind::hipMemcpyDeviceToDevice));
    
    cuAssert(cudnnDropoutForward(gpuPrm->cudnn,
        gpuPrm->drop_desc,
        gpuPrm->in_desc,
        gpuPrm->in_mem,
        gpuPrm->out_desc,
        inout,
        gpuPrm->d_ws,
        gpuPrm->wsSz));
        
}

void dropOutBackward(SN_Base::snFloat dropOut, SN_Base::snFloat* inout, const SN_Base::snSize& outsz, uint32_t deviceId, void** pGpuPrm){

    hipSetDevice(deviceId);

    dropOutInit(pGpuPrm, dropOut, inout, outsz);

    gpuParams* gpuPrm = (gpuParams*)*pGpuPrm;

    cuAssert(hipMemcpy(gpuPrm->in_mem, inout, outsz.size() * sizeof(snFloat), hipMemcpyKind::hipMemcpyDeviceToDevice));

    cuAssert(cudnnDropoutBackward(gpuPrm->cudnn,
        gpuPrm->drop_desc,
        gpuPrm->in_desc,
        gpuPrm->in_mem,
        gpuPrm->out_desc,
        inout,
        gpuPrm->d_ws,
        gpuPrm->wsSz));

}