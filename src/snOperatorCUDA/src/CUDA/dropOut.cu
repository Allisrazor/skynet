#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hiprand.h>

#include "../stdafx.h"

using namespace SN_Base;

__global__ void dropOutLern(SN_Base::snFloat dropOut, const snSize& outsz, snFloat* rnd, SN_Base::snFloat* out){
    
    size_t outStepByD = outsz.w * outsz.h,     // step out by input
           outStepByN = outStepByD * outsz.d;  // step out by batch       

    // gridDim.x - number of out layers
    // gridDim.y - batch size

    out += blockIdx.x * outStepByD + blockIdx.y * outStepByN;

    unsigned int i = threadIdx.x;
    
    if (rnd[i] < dropOut){

        while (i < outStepByD){

            out[i] = 0.F;

            i += blockDim.x;
        }
    }
}

__global__ void dropOutInf(SN_Base::snFloat dropOut, const snSize& outsz, snFloat* out){

    size_t outStepByD = outsz.w * outsz.h,     // step out by input
           outStepByN = outStepByD * outsz.d;  // step out by batch       

    // gridDim.x - number of out layers
    // gridDim.y - batch size

    out += blockIdx.x * outStepByD + blockIdx.y * outStepByN;

    unsigned int i = threadIdx.x;

    while (i < outStepByD){

        out[i] *= (1.F - dropOut);

        i += blockDim.x;
    }
}


void dropOut(bool isLern, snFloat dropOut, const snSize& outsz, snFloat* inout){
       
    if (isLern){
        int blockSz = 128;

        float* rndData = nullptr;
        cuAssert(hipMalloc((void**)&rndData, blockSz * sizeof(float)));

        hiprandGenerator_t gen;
        cuAssert(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));

        cuAssert(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));

        cuAssert(hiprandGenerateUniform(gen, rndData, blockSz));

        dim3 dimBlock(blockSz);
        dim3 dimGrid(int(outsz.d), int(outsz.n));
               
        dropOutLern << < dimGrid, dimBlock >> >(dropOut, outsz, rndData, inout);
        
        cuAssert(hiprandDestroyGenerator(gen));
        cuAssert(hipFree(rndData));
    }
    else{
     
        dim3 dimBlock(128);
        dim3 dimGrid(int(outsz.d), int(outsz.n));
        
        dropOutInf << <dimGrid, dimBlock >> >(dropOut, outsz, inout);
    }
}