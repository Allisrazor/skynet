
#include <hip/hip_runtime.h>
#include <hipDNN.h>

#include "../stdafx.h"

using namespace SN_Base;

void dropOutForward(SN_Base::snFloat dropOut, SN_Base::snFloat* inout, const SN_Base::snSize& iosz, uint32_t deviceId){
       
    hipSetDevice(deviceId);

            
}

void dropOutBackward(SN_Base::snFloat* inout, const SN_Base::snSize& iosz, uint32_t deviceId){

    hipSetDevice(deviceId);
 
   

}