
#include <hip/hip_runtime.h>
//
// SkyNet Project
// Copyright (C) 2018 by Contributors <https://github.com/Tyill/skynet>
//
// This code is licensed under the MIT License.
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files(the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and / or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions :
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
// THE SOFTWARE.
//

#ifdef SN_CUDA

#include <cublas_v2.h>
#include <cuda_runtime.h>
#include "../stdafx.h"
#include "SNOperator/src/Operator/fullyConnected.h"

using namespace std;
using namespace SN_Base;
         
#ifndef cuCHECK
#define cuCHECK(func) if (func != 0){ ERROR_MESS("CUDA error: " + cudaGetErrorString(cudaGetLastError())); return;}
#endif

void FullyConnected::iniParamCUDA(const snSize& insz, size_t kernel, map<string, void*>& gpuPrm){
    cudaSetDevice(gpuDeviceId_);

    size_t ida = insz.w * insz.h * insz.d, bsz = insz.n;

    if (gpuPrm.find("hcuBLAS") == gpuPrm.end()){
        
        cublasHandle_t cuHandle = nullptr;
        cuCHECK(cublasCreate(&cuHandle));

        gpuPrm["hcuBLAS"] = cuHandle;
          
        gpuPrm["d_in"] = 0;
        gpuPrm["d_w"] = 0;
        gpuPrm["d_out"] = 0;
        gpuPrm["d_grout"] = 0;
        gpuPrm["d_dw"] = 0;

        if (!gpuClearMem_){
            cuCHECK(cudaMalloc(&gpuPrm["d_in"], bsz * ida * sizeof(snFloat)));
            cuCHECK(cudaMalloc(&gpuPrm["d_w"], (ida + 1) * kernel * sizeof(snFloat)));
            cuCHECK(cudaMalloc(&gpuPrm["d_out"], bsz * kernel * sizeof(snFloat)));
            cuCHECK(cudaMalloc(&gpuPrm["d_grout"], bsz * ida * sizeof(snFloat)));
            cuCHECK(cudaMalloc(&gpuPrm["d_dw"], (ida + 1) * kernel * sizeof(snFloat)));
        }
    }
    else if (!gpuClearMem_){
          
        cuCHECK(cudaFree(gpuPrm["d_in"]));    gpuPrm["d_in"] = 0;
        cuCHECK(cudaFree(gpuPrm["d_w"]));     gpuPrm["d_w"] = 0;
        cuCHECK(cudaFree(gpuPrm["d_out"]));   gpuPrm["d_out"] = 0;
        cuCHECK(cudaFree(gpuPrm["d_grout"])); gpuPrm["d_grout"] = 0;
        cuCHECK(cudaFree(gpuPrm["d_dw"]));    gpuPrm["d_dw"] = 0;

        cuCHECK(cudaMalloc(&gpuPrm["d_in"], bsz * ida * sizeof(snFloat)));
        cuCHECK(cudaMalloc(&gpuPrm["d_w"], (ida + 1) * kernel * sizeof(snFloat)));
        cuCHECK(cudaMalloc(&gpuPrm["d_out"], bsz * kernel * sizeof(snFloat)));
        cuCHECK(cudaMalloc(&gpuPrm["d_grout"], bsz * ida * sizeof(snFloat)));
        cuCHECK(cudaMalloc(&gpuPrm["d_dw"], (ida + 1) * kernel * sizeof(snFloat)));
    }
}
         
void FullyConnected::freeParamCUDA(map<string, void*>& gpuPrm){
    cudaSetDevice(gpuDeviceId_);

    if (gpuPrm.find("hcuBLAS") == gpuPrm.end()) return;

    cublasDestroy((cublasHandle_t)gpuPrm["hcuBLAS"]);

    if (!gpuClearMem_){
        for (auto p : gpuPrm)
            if (p.first != "hcuBLAS")  cudaFree(p.second);
    }
}

__global__ void cuFwdBias(size_t kernel, snSize insz, snFloat* weight, snFloat* output){
       
    weight += insz.w * insz.h * insz.d * kernel;
   
    snFloat* out = output + kernel * blockIdx.x;
    unsigned int k = threadIdx.x;
    while (k < kernel){

        out[k] += weight[k];

        k += blockDim.x;
    }   
}

void FullyConnected::forwardCUDA(size_t kernel, const snSize& insz, snFloat* input, snFloat* weight, snFloat* output, map<string, void*>& gpuPrm){
    cudaSetDevice(gpuDeviceId_);

    if (gpuPrm.find("hcuBLAS") == gpuPrm.end()) return;

    cublasHandle_t hcuBLAS = (cublasHandle_t)gpuPrm["hcuBLAS"];

    int ida = int(insz.w * insz.h * insz.d), bsz = int(insz.n), krn = int(kernel);
   
    snFloat *d_in  = (snFloat*)gpuPrm["d_in"],
            *d_w   = (snFloat*)gpuPrm["d_w"], 
            *d_out = (snFloat*)gpuPrm["d_out"];
   
    if (gpuClearMem_){
        cuCHECK(cudaMalloc(reinterpret_cast<void**>(&d_in), bsz * ida * sizeof(snFloat)));
        cuCHECK(cudaMalloc(reinterpret_cast<void**>(&d_w), (ida + 1) * kernel * sizeof(snFloat)));
        cuCHECK(cudaMalloc(reinterpret_cast<void**>(&d_out), bsz * kernel * sizeof(snFloat)));
    }

    cuCHECK(cublasSetMatrix(bsz, ida, sizeof(snFloat), input, bsz, d_in, bsz));
  
    cuCHECK(cublasSetMatrix(ida, krn, sizeof(snFloat), weight, ida, d_w, ida));
   
    // Out = α * W * In + βC
    // In - data input matrix - values from the previous layer
    // W - weights matrix
    // Out - output matrix
    float alpha = 1.0f, beta = 0.0f;
    cuCHECK(cublasSgemm(hcuBLAS,
        CUBLAS_OP_N,
        CUBLAS_OP_N,
        krn,                           // W, cols
        bsz,                           // In, rows
        ida,                           // In, cols, В М - rows            
        &alpha,                        // α
        d_w,                           // W
        krn,                           // W, step to next W (W21 - W11)
        d_in,                          // In
        ida,                           // In, step to next X (X21 - X11)  
        &beta,                         // β
        d_out,                         // Out
        krn));                         // Out, step to next Y (Y21 - Y11) 
    
    // +bias
    cuFwdBias <<< insz.n, 128 >>> (kernel, insz, d_w, d_out);

    // result
    cuCHECK(cublasGetMatrix(bsz, krn, sizeof(snFloat), d_out, bsz, output, bsz));
    
    if (gpuClearMem_){
        cuCHECK(cudaFree(d_in));
        cuCHECK(cudaFree(d_w));
        cuCHECK(cudaFree(d_out));
    }
}

__global__ void cuBwdBias(size_t kernel, snSize insz, snFloat* gradIn, snFloat* dWOut){
    
    // bias
    dWOut += insz.w * insz.h * insz.d * kernel;
    unsigned int k = threadIdx.x;
    while (k < kernel){
   
        snFloat* grin = gradIn + k, b = 0;
        for (size_t j = 0; j < insz.n; ++j)
            b += grin[kernel * j];

        dWOut[k] = b;
        k += blockDim.x;
    }
}

void FullyConnected::backwardCUDA_GW(size_t kernel, snFloat* weight,
    const snSize& insz, snFloat* input, snFloat* gradIn, snFloat* gradOut, snFloat* dWOut, map<string, void*>& gpuPrm){
    cudaSetDevice(gpuDeviceId_);

    if (gpuPrm.find("hcuBLAS") == gpuPrm.end()) return;

    cublasHandle_t hcuBLAS = (cublasHandle_t)gpuPrm["hcuBLAS"];

    int ida = int(insz.w * insz.h * insz.d), bsz = int(insz.n), krn = int(kernel);

    snFloat* d_grin = (snFloat*)gpuPrm["d_out"],
           * d_in = (snFloat*)gpuPrm["d_in"],
           * d_w = (snFloat*)gpuPrm["d_w"],
           * d_dw = (snFloat*)gpuPrm["d_dw"],
           * d_grout = (snFloat*)gpuPrm["d_grout"];

    if (gpuClearMem_){
        cuCHECK(cudaMalloc(&d_in, bsz * ida * sizeof(snFloat)));          
        cuCHECK(cudaMalloc(&d_w, (ida + 1) * kernel * sizeof(snFloat)));        
        cuCHECK(cudaMalloc(&d_grin, bsz * kernel * sizeof(snFloat)));    
        cuCHECK(cudaMalloc(&d_grout, bsz * ida * sizeof(snFloat)));
        cuCHECK(cudaMalloc(&d_dw, (ida + 1) * kernel * sizeof(snFloat)));
    }

    cuCHECK(cublasSetMatrix(bsz, ida, sizeof(snFloat), input, bsz, d_in, bsz));
  
    cuCHECK(cublasSetMatrix(bsz, krn, sizeof(snFloat), gradIn, bsz, d_grin, bsz));

    // Weight gradient
    // dW = αIn^T * GrIn + βdW
    // In - data input matrix from previous layer
    // GrIn - gradient matrix from the next layer
    float alpha = 1.0F / insz.n, beta = 0.0f;
    cuCHECK(cublasSgemm(hcuBLAS,
        CUBLAS_OP_N,
        CUBLAS_OP_T,
        krn,                     // GrIn, cols
        ida,                     // In, cols (+1 - X0)      
        bsz,                     // In, rows
        &alpha,                  // α                
        d_grin,                  // GrIn
        krn,                     // GrIn, step to next 
        d_in,                    // In
        ida,                     // In, step to next  X (X21 - X11)  
        &beta,                   // β               
        d_dw,                    // dW            
        krn));                   // dW, step to next 

    // bias
    cuBwdBias <<< 1, 128 >>> (kernel, insz, d_grin, d_dw);

    cuCHECK(cublasGetMatrix(ida, krn, sizeof(snFloat), d_dw, ida, dWOut, ida));
     
    cuCHECK(cudaMemcpy(output, gpuPrm->d_out, outsz.size() * sizeof(snFloat), cudaMemcpyDeviceToHost));


    cuCHECK(cublasSetMatrix(ida, krn, sizeof(snFloat), weight, ida, d_w, ida));

    //// Gradient for previous layer
    //// GrOut = αGrIn * W^T + βGrOut
    //// GrIn - gradient matrix from the next layer
    //// W - weight
    alpha = 1.F;
    cuCHECK(cublasSgemm(hcuBLAS,
        CUBLAS_OP_T,
        CUBLAS_OP_N,
        ida - 1,                 // W, cols (+1 - X0)     
        bsz,                     // W, rows
        krn,                     // GrIn, cols
        &alpha,                  // α                               
        d_w,                     // W
        krn,                     // W, step to next 
        d_grin,                  // GrIn
        krn,                     // GrIn, step to next 
        &beta,                   // β               
        d_grout,                 // GrOut                                  
        ida - 1));               // GrOut, step to next 
     
   
    // result
    cuCHECK(cublasGetMatrix(bsz, ida - 1, sizeof(snFloat), d_grout, bsz, gradOut, bsz));
 
    if (gpuClearMem_){
        cuCHECK(cudaFree(d_in));
        cuCHECK(cudaFree(d_w));
        cuCHECK(cudaFree(d_grin));
        cuCHECK(cudaFree(d_grout));
        cuCHECK(cudaFree(d_dw));
    }
}

void FullyConnected::backwardCUDA_G(size_t kernel, snFloat* weight, const snSize& insz, snFloat* gradIn, snFloat* gradOut, map<string, void*>& gpuPrm){
    cudaSetDevice(gpuDeviceId_);

    if (gpuPrm.find("hcuBLAS") == gpuPrm.end()) return;

    cublasHandle_t hcuBLAS = (cublasHandle_t)gpuPrm["hcuBLAS"];

    int ida = int(insz.w * insz.h * insz.d + 1), bsz = int(insz.n), krn = int(kernel);

    snFloat* d_grin = (snFloat*)gpuPrm["d_out"],
           * d_w = (snFloat*)gpuPrm["d_w"],
           * d_grout = (snFloat*)gpuPrm["d_grout"];

    if (gpuClearMem_){
        cuCHECK(cudaMalloc(reinterpret_cast<void**>(&d_w), ida * kernel * sizeof(snFloat)));
        cuCHECK(cudaMalloc(reinterpret_cast<void**>(&d_grin), bsz * kernel * sizeof(snFloat)));
        cuCHECK(cudaMalloc(reinterpret_cast<void**>(&d_grout), bsz * (ida - 1) * sizeof(snFloat)));
    }

    cuCHECK(cublasSetMatrix(bsz, krn, sizeof(snFloat), gradIn, bsz, d_grin, bsz));

    cuCHECK(cublasSetMatrix(ida - 1, krn, sizeof(snFloat), weight + kernel, ida - 1, d_w, ida - 1));

    //// Gradient for previous layer
    //// GrOut = αGrIn * W^T + βGrOut
    //// GrIn - gradient matrix from the next layer
    //// W - weight
    float alpha = 1.0F, beta = 0.0f;
    cuCHECK(cublasSgemm(hcuBLAS,
        CUBLAS_OP_T,
        CUBLAS_OP_N,
        ida - 1,                 // W, cols (+1 - X0)     
        bsz,                     // W, rows
        krn,                     // GrIn, cols
        &alpha,                  // α                               
        d_w,                     // W
        krn,                     // W, step to next 
        d_grin,                  // GrIn
        krn,                     // GrIn, step to next 
        &beta,                   // β         
        d_grout,                 // GrOut                          
        ida - 1));               // GrOut, step to next 

    cuCHECK(cublasGetMatrix(bsz, ida - 1, sizeof(snFloat), d_grout, bsz, gradOut, bsz));

    if (gpuClearMem_){
        cuCHECK(cudaFree(d_w));
        cuCHECK(cudaFree(d_grin));
        cuCHECK(cudaFree(d_grout));
    }
}

#endif 