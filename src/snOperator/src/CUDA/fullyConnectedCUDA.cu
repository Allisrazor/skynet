
#include <hip/hip_runtime.h>
//
// SkyNet Project
// Copyright (C) 2018 by Contributors <https://github.com/Tyill/skynet>
//
// This code is licensed under the MIT License.
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files(the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and / or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions :
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
// THE SOFTWARE.
//

#ifdef SN_CUDA

#include <cublas_v2.h>
#include <cuda_runtime.h>
#include "../stdafx.h"
#include "SNOperator/src/Operator/fullyConnected.h"

using namespace std;
using namespace SN_Base;
          
void FullyConnected::iniParamCUDA(snSize insz, size_t kernel, map<string, void*>& gpuPrm){
    
    size_t ida = insz.w * insz.h * insz.d + 1, bsz = insz.n;

    if (gpuPrm.find("hcuBLAS") == gpuPrm.end()){
        
        cublasHandle_t cuHandle = nullptr;
        int sts = cublasCreate(&cuHandle);
        if (sts != CUBLAS_STATUS_SUCCESS){
            ERROR_MESS("fwdFullyConnected CUBLAS initialization error: sts " + to_string(sts));
            return;
        }
        gpuPrm["hcuBLAS"] = cuHandle;
                            
        snFloat* d_in_fwd = 0, *d_w_fwd = 0, *d_out_fwd = 0;
        cudaMalloc(reinterpret_cast<void**>(&d_in_fwd),     bsz * ida * sizeof(snFloat)); gpuPrm["d_in_fwd"]  = d_in_fwd;
        cudaMalloc(reinterpret_cast<void**>(&d_w_fwd),   ida * kernel * sizeof(snFloat)); gpuPrm["d_w_fwd"]   = d_w_fwd;
        cudaMalloc(reinterpret_cast<void**>(&d_out_fwd), bsz * kernel * sizeof(snFloat)); gpuPrm["d_out_fwd"] = d_out_fwd;
         
        if (batchNormType_ != batchNormType::none){
            snFloat* d_in_bn = 0, *d_out_bn = 0, *d_norm_bn = 0, *d_mean_bn = 0,
                *d_varce_bn = 0, *d_scale_bn = 0, *d_dScale_bn = 0, *d_schift_bn = 0, *d_dSchift_bn = 0, *d_onc_bn = 0;

            cudaMalloc(reinterpret_cast<void**>(&d_in_bn),   bsz * kernel * sizeof(snFloat)); gpuPrm["d_in_bn"]      = d_in_bn;
            cudaMalloc(reinterpret_cast<void**>(&d_out_bn),  bsz * kernel * sizeof(snFloat)); gpuPrm["d_out_bn"]     = d_out_bn;
            cudaMalloc(reinterpret_cast<void**>(&d_norm_bn), bsz * kernel * sizeof(snFloat)); gpuPrm["d_norm_bn"]    = d_norm_bn;
            cudaMalloc(reinterpret_cast<void**>(&d_mean_bn),       kernel * sizeof(snFloat)); gpuPrm["d_mean_bn"]    = d_mean_bn;
            cudaMalloc(reinterpret_cast<void**>(&d_varce_bn),      kernel * sizeof(snFloat)); gpuPrm["d_varce_bn"]   = d_varce_bn;
            cudaMalloc(reinterpret_cast<void**>(&d_scale_bn),      kernel * sizeof(snFloat)); gpuPrm["d_scale_bn"]   = d_scale_bn;
            cudaMalloc(reinterpret_cast<void**>(&d_dScale_bn),     kernel * sizeof(snFloat)); gpuPrm["d_dScale_bn"]  = d_dScale_bn;
            cudaMalloc(reinterpret_cast<void**>(&d_schift_bn),     kernel * sizeof(snFloat)); gpuPrm["d_schift_bn"]  = d_schift_bn;
            cudaMalloc(reinterpret_cast<void**>(&d_dSchift_bn),    kernel * sizeof(snFloat)); gpuPrm["d_dSchift_bn"] = d_dSchift_bn;
            cudaMalloc(reinterpret_cast<void**>(&d_onc_bn),           bsz * sizeof(snFloat)); gpuPrm["d_onc_bn"]     = d_onc_bn;
        }
    }
    else{
        snFloat* d_in_fwd  = (snFloat*)gpuPrm["d_in_fwd"],
               * d_w_fwd   = (snFloat*)gpuPrm["d_w_fwd"],
               * d_out_fwd = (snFloat*)gpuPrm["d_out_fwd"];            

        cudaFree(d_in_fwd);  cudaMalloc(reinterpret_cast<void**>(&d_in_fwd),     bsz * ida * sizeof(snFloat)); gpuPrm["d_in_fwd"]  = d_in_fwd;
        cudaFree(d_w_fwd);   cudaMalloc(reinterpret_cast<void**>(&d_w_fwd),   ida * kernel * sizeof(snFloat)); gpuPrm["d_w_fwd"]   = d_w_fwd;
        cudaFree(d_out_fwd); cudaMalloc(reinterpret_cast<void**>(&d_out_fwd), bsz * kernel * sizeof(snFloat)); gpuPrm["d_out_fwd"] = d_out_fwd;

        if (batchNormType_ != batchNormType::none){
            snFloat* d_in_bn   = (snFloat*)gpuPrm["d_in_bn"],
                   * d_out_bn  = (snFloat*)gpuPrm["d_out_bn"],
                   * d_norm_bn = (snFloat*)gpuPrm["d_norm_bn"],
                   * d_onc_bn  = (snFloat*)gpuPrm["d_onc_bn"];

            cudaFree(d_in_bn);   cudaMalloc(reinterpret_cast<void**>(&d_in_bn),   bsz * kernel * sizeof(snFloat)); gpuPrm["d_in_bn"]   = d_in_bn;
            cudaFree(d_out_bn);  cudaMalloc(reinterpret_cast<void**>(&d_out_bn),  bsz * kernel * sizeof(snFloat)); gpuPrm["d_out_bn"]  = d_out_bn;
            cudaFree(d_norm_bn); cudaMalloc(reinterpret_cast<void**>(&d_norm_bn), bsz * kernel * sizeof(snFloat)); gpuPrm["d_norm_bn"] = d_norm_bn;
            cudaFree(d_onc_bn);  cudaMalloc(reinterpret_cast<void**>(&d_onc_bn),           bsz * sizeof(snFloat)); gpuPrm["d_onc_bn"]  = d_onc_bn;
        } 
    }
}
         
void FullyConnected::freeParamCUDA(map<string, void*>& gpuPrm){
    
    if (gpuPrm.find("hcuBLAS") == gpuPrm.end()) return;

    cublasDestroy((cublasHandle_t)gpuPrm["hcuBLAS"]);

    for (auto p : gpuPrm)
        if (p.first != "hcuBLAS")  cudaFree(p.second);
}

void FullyConnected::forwardCUDA(size_t kernel, snSize insz, snFloat* input, snFloat* weight, snFloat* output, map<string, void*>& gpuPrm){
      
    if (gpuPrm.find("hcuBLAS") == gpuPrm.end()) return;

    size_t ida = insz.w * insz.h * insz.d + 1, bsz = insz.n;
   
    snFloat *d_in  = (snFloat*)gpuPrm["d_in_FWD"],
            *d_w   = (snFloat*)gpuPrm["d_w_FWD"], 
            *d_out = (snFloat*)gpuPrm["d_out_FWD"];
   
    cublasSetMatrix(bsz, ida, sizeof(snFloat), input, bsz, d_in, bsz);
    
    cublasSetMatrix(ida, kernel, sizeof(snFloat), weight, ida, d_w, ida);
   
    // Out = α * W * In + βC
    // In - матрица вход данных - значения с предыд слоя
    // W - матрица весов
    // Out - матрица выход данных
    float alpha = 1.0f, beta = 0.0f;
    cublasSgemm((cublasHandle_t)gpuPrm["hcuBLAS"],
        CUBLAS_OP_N,
        CUBLAS_OP_N,
        kernel,                        // W, столбцов, кол-во скрытых нейронов 
        bsz,                           // In, строк, кол-во изобр в батче
        ida,                           // In, столбцов, В М - строк, кол-во вх нейронов - размер одного изображения из батча. (+1 - X0)                   
        &alpha,                        // α, коэф
        d_w,                           // W, веса
        kernel,                        // W, шаг до след W (W21 - W11)
        d_in,                          // In, вх данные - нейроны пришедшие с предыд слоя
        ida,                           // In, шаг до след X (X21 - X11)  
        &beta,                         // β, коэф
        d_out,                         // Out, выходные данные - нейроны для след слоя
        kernel);                       // Out, шаг до след Y (Y21 - Y11) 
    
    cublasGetMatrix(bsz, kernel, sizeof(snFloat), d_out, bsz, output, bsz); 
   
}

void FullyConnected::backwardCUDA_GW(size_t kernel, snFloat* weight,
    snSize insz, snFloat* input, snFloat* gradIn, snFloat* gradOut, snFloat* dWOut, map<string, void*>&){


}

void FullyConnected::backwardCUDA_G(size_t kernel, snFloat* weight, snSize insz, snFloat* gradIn, snFloat* gradOut, map<string, void*>&){


}


#endif 


// SN_CUDA

//
//#define m 3 // a - mxk matrix
//#define n 4 // b - kxn matrix
//#define k 5 // c - mxn matrix
//
//
//
//cudaError_t cudaStat; // cudaMalloc status
//cublasStatus_t stat; // CUBLAS functions status
//cublasHandle_t handle; // CUBLAS context   
//
//float* a = (float*)malloc(m*k* sizeof(float)); // host memory for a
//float* b = (float*)malloc(k*n* sizeof(float)); // host memory for b
//float* c = (float*)malloc(m*n* sizeof(float)); // host memory for c
//
//// define an mxk matrix a column by column
//printf("a:\n");
//int ind = 0;                             // a:
//for (int i = 0; i < m; ++i){                   // 0  1  2  3  4
//    for (int j = 0; j < k; ++j){               // 5  6  7  8  9
//        a[j + i * k] = (float)++ind;           // 10 11 12 13 14
//        printf(" %5.0f", a[j + i * k]);
//    }
//    printf("\n");
//}
//
//// define a kxn matrix b column by column   
//printf("b:\n");
//ind = 1;                                 // b: 
//for (int i = 0; i < k; ++i){                   // 1  2  3  4
//    for (int j = 0; j < n; ++j){               // 5  6  7  8
//        b[j + i * n] = (float)++ind;           // 9  10 11 12
//        printf(" %5.0f", b[j + i * n]);       // 13 14 15 16
//    }                                          // 17 18 19 20 
//    printf("\n");
//}
//
//// on the device
//float * d_a; // d_a - a on the device
//float * d_b; // d_b - b on the device
//float * d_c; // d_c - c on the device
//cudaStat = cudaMalloc((void **)& d_a, m*k* sizeof(*a)); // device
//
//// memory alloc for a
//cudaStat = cudaMalloc((void **)& d_b, k*n* sizeof(*b)); // device
//
//// memory alloc for b
//cudaStat = cudaMalloc((void **)& d_c, m*n* sizeof(*c)); // device
//
//// memory alloc for c
//stat = cublasCreate(&handle); // initialize CUBLAS context
//
//// copy matrices from the host to the device
//stat = cublasSetMatrix(m, k, sizeof(*a), a, m, d_a, m); //a -> d_a
//stat = cublasSetMatrix(k, n, sizeof(*b), b, k, d_b, k); //b -> d_b
//stat = cublasSetMatrix(m, n, sizeof(*c), c, m, d_c, m); //c -> d_c
//
//// a - mxk matrix
//// b - kxn matrix
//// c - mxn matrix
//
//float al = 1.0f;   // al =1
//float bet = 0.0f;  // bet =1
//stat = cublasSgemm(handle,
//    CUBLAS_OP_N,
//    CUBLAS_OP_N,
//    n,   // строки 1й
//    m,   // столбцы 2й
//    k,   // столбцы 1й
//    &al,
//    d_b,
//    n,
//    d_a,
//    k,
//    &bet,
//    d_c,
//    n);
//
//stat = cublasGetMatrix(m, n, sizeof(*c), d_c, m, c, m); // cp d_c - >c
//printf("c after Sgemm :\n");
//for (int i = 0; i < m; i++){
//    for (int j = 0; j < n; j++){
//        printf(" %7.0f", c[j + i * n]);
//    }
//    printf("\n");
//}
//
//bool ff = false;
//#undef m
//#undef n
//#undef k