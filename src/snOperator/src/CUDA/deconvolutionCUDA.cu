
#include <hip/hip_runtime.h>
//
// SkyNet Project
// Copyright (C) 2018 by Contributors <https://github.com/Tyill/skynet>
//
// This code is licensed under the MIT License.
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files(the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and / or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions :
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
// THE SOFTWARE.
//

#ifdef SN_CUDNN

#include <cuda_runtime.h>
#include <cudnn.h>
#include "../stdafx.h"
#include "snOperator/src/Operator/deconvolution.h"

using namespace std;
using namespace SN_Base;

#ifndef cuCHECK
#define cuCHECK(func) if (func != 0){ ERROR_MESS("CUDA error: " + cudaGetErrorString(cudaGetLastError())); return;}
#endif

struct gpuParams{

    cudnnHandle_t cudnn = 0;
    cudnnConvolutionDescriptor_t conv_desc = 0;
    cudnnTensorDescriptor_t in_desc = 0;
    cudnnTensorDescriptor_t out_desc = 0;
    cudnnTensorDescriptor_t grin_desc = 0;
    cudnnTensorDescriptor_t grout_desc = 0;
    cudnnFilterDescriptor_t w_desc = 0;
    cudnnFilterDescriptor_t dw_desc = 0;
    cudnnTensorDescriptor_t bias_desc = 0;

    cudnnConvolutionFwdAlgo_t algoFwd;
    cudnnConvolutionBwdDataAlgo_t algoBwdData;
    cudnnConvolutionBwdFilterAlgo_t algoBwdW;

    size_t wsFwdSz = 0;
    size_t wsBwdDataSz = 0;
    size_t wsBwdWSz = 0;
    size_t inszMem = 0;

    snFloat* d_in = 0;
    snFloat* d_w = 0;
    snFloat* d_dw = 0;
    snFloat* d_bias = 0;
    snFloat* d_out = 0;
    snFloat* d_grout = 0;
    void* d_wsFwd = 0;
    void* d_wsBwdData = 0;
    void* d_wsBwdW = 0;

};

void Deconvolution::iniParamCUDA(bool isLern, const snSize& insz, const snSize& outsz,
    const deconvParams& prms, void** pGpuPrm){

    cudaSetDevice(gpuDeviceId_);

    bool isFirst = false;

    gpuParams* gpuPrm = (gpuParams*)*pGpuPrm;
    if (!gpuPrm){

        cudaDeviceProp cu_deviceProps;

        cudaGetDeviceProperties(&cu_deviceProps, 0);
        if (cu_deviceProps.major < 3){
            ERROR_MESS("%s requires SM >= 3.0");
            return;
        }
        gpuPrm = new gpuParams();
        memset(gpuPrm, 0, sizeof(gpuParams));
        *pGpuPrm = gpuPrm;

        cudnnHandle_t cudnn = nullptr;
        cuCHECK(cudnnCreate(&cudnn));
        gpuPrm->cudnn = cudnn;

        isFirst = true;
    }
    
    // input
    cudnnTensorDescriptor_t in_desc = nullptr;
    cuCHECK(cudnnCreateTensorDescriptor(&in_desc));
    cuCHECK(cudnnSetTensor4dDescriptor(in_desc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, int(insz.n), int(insz.d), int(insz.h), int(insz.w)));
    if (!isFirst)
        cuCHECK(cudnnDestroyTensorDescriptor((cudnnTensorDescriptor_t)gpuPrm->in_desc));
    gpuPrm->in_desc = in_desc;
      
    // w      
    cudnnFilterDescriptor_t w_desc = nullptr;
    cuCHECK(cudnnCreateFilterDescriptor(&w_desc));
    cuCHECK(cudnnSetFilter4dDescriptor(w_desc, CUDNN_DATA_FLOAT, CUDNN_TENSOR_NCHW,
        int(insz.d), int(outsz.d), int(prms.fHeight), int(prms.fWidth)));
    if (!isFirst)
        cuCHECK(cudnnDestroyFilterDescriptor((cudnnFilterDescriptor_t)gpuPrm->w_desc));
    gpuPrm->w_desc = w_desc;
     
    // conv
    cudnnConvolutionDescriptor_t conv_desc = nullptr;
    cuCHECK(cudnnCreateConvolutionDescriptor(&conv_desc));
    cuCHECK(cudnnSetConvolution2dDescriptor(conv_desc, 0, 0, int(prms.stride), int(prms.stride), 1, 1,
        CUDNN_CROSS_CORRELATION, CUDNN_DATA_FLOAT));
    if (!isFirst)
        cuCHECK(cudnnDestroyConvolutionDescriptor((cudnnConvolutionDescriptor_t)gpuPrm->conv_desc));
    gpuPrm->conv_desc = conv_desc;

    // output
    cudnnTensorDescriptor_t out_desc;
    cuCHECK(cudnnCreateTensorDescriptor(&out_desc));
    cuCHECK(cudnnSetTensor4dDescriptor(out_desc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT,
        int(outsz.n), int(outsz.d), int(outsz.h), int(outsz.w)));
    if (!isFirst)
        cuCHECK(cudnnDestroyTensorDescriptor((cudnnTensorDescriptor_t)gpuPrm->out_desc));
    gpuPrm->out_desc = out_desc;

    // algorithm
    cudnnConvolutionBwdDataAlgo_t algoBwdData;
    cuCHECK(cudnnGetConvolutionBackwardDataAlgorithm(gpuPrm->cudnn, w_desc, in_desc, conv_desc, out_desc,
        CUDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &algoBwdData));
    gpuPrm->algoBwdData = algoBwdData;
  
    // workspace
    size_t wsBwdDataSz = 0;
    cuCHECK(cudnnGetConvolutionBackwardDataWorkspaceSize(gpuPrm->cudnn, w_desc, in_desc, conv_desc, out_desc, algoBwdData, &wsBwdDataSz));
    gpuPrm->wsBwdDataSz = wsBwdDataSz;


    size_t wsFwdSz = 0, wsBwdWSz = 0;
    if (isLern){
        // grin
        cudnnTensorDescriptor_t grin_desc;
        cuCHECK(cudnnCreateTensorDescriptor(&grin_desc));
        cuCHECK(cudnnSetTensor4dDescriptor(grin_desc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT,
            int(outsz.n), int(outsz.d), int(outsz.h), int(outsz.w)));
        if (!isFirst)
            cuCHECK(cudnnDestroyTensorDescriptor((cudnnTensorDescriptor_t)gpuPrm->grin_desc));
        gpuPrm->grin_desc = grin_desc;

        // grout
        cudnnTensorDescriptor_t grout_desc;
        cuCHECK(cudnnCreateTensorDescriptor(&grout_desc));
        cuCHECK(cudnnSetTensor4dDescriptor(grout_desc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, int(insz.n), int(insz.d), int(insz.h), int(insz.w)));
        if (!isFirst)
            cuCHECK(cudnnDestroyTensorDescriptor((cudnnTensorDescriptor_t)gpuPrm->grout_desc));
        gpuPrm->grout_desc = grout_desc;

        // dw     
        cudnnFilterDescriptor_t dw_desc = nullptr;
        cuCHECK(cudnnCreateFilterDescriptor(&dw_desc));
        cuCHECK(cudnnSetFilter4dDescriptor(dw_desc, CUDNN_DATA_FLOAT, CUDNN_TENSOR_NCHW,
            int(insz.d), int(outsz.d), int(prms.fHeight), int(prms.fWidth)));
        if (!isFirst)
            cuCHECK(cudnnDestroyFilterDescriptor((cudnnFilterDescriptor_t)gpuPrm->dw_desc));
        gpuPrm->dw_desc = dw_desc;

        // bias
        cudnnTensorDescriptor_t bias_desc;
        cuCHECK(cudnnCreateTensorDescriptor(&bias_desc));
        cuCHECK(cudnnSetTensor4dDescriptor(bias_desc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT,
            1, int(insz.d), 1, 1));
        if (!isFirst)
            cuCHECK(cudnnDestroyTensorDescriptor((cudnnTensorDescriptor_t)gpuPrm->bias_desc));
        gpuPrm->bias_desc = bias_desc;

        // algorithm
        cudnnConvolutionFwdAlgo_t algoFwd;
        cuCHECK(cudnnGetConvolutionForwardAlgorithm(gpuPrm->cudnn, grin_desc, w_desc, conv_desc, grout_desc,
            CUDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algoFwd));
        gpuPrm->algoFwd = algoFwd;

        cudnnConvolutionBwdFilterAlgo_t algoBwdW;
        cuCHECK(cudnnGetConvolutionBackwardFilterAlgorithm(gpuPrm->cudnn, grin_desc, in_desc, conv_desc, dw_desc,
            CUDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &algoBwdW));
        gpuPrm->algoBwdW = algoBwdW;

        // workspace       
        cuCHECK(cudnnGetConvolutionForwardWorkspaceSize(gpuPrm->cudnn, grin_desc, w_desc, conv_desc, grout_desc, algoFwd, &wsFwdSz));
        gpuPrm->wsFwdSz = wsFwdSz;
                
        cuCHECK(cudnnGetConvolutionBackwardFilterWorkspaceSize(gpuPrm->cudnn, grin_desc, in_desc, conv_desc, dw_desc, algoBwdW, &wsBwdWSz));
        gpuPrm->wsBwdWSz = wsBwdWSz;
    }

    if (isFirst && !gpuClearMem_){
        cuCHECK(cudaMalloc(&gpuPrm->d_in, insz.size() * sizeof(snFloat)));
        cuCHECK(cudaMalloc(&gpuPrm->d_w, prms.fWidth * prms.fHeight * insz.d * outsz.d * sizeof(snFloat)));
        cuCHECK(cudaMalloc(&gpuPrm->d_out, outsz.size() * sizeof(snFloat)));
        cuCHECK(cudaMalloc(&gpuPrm->d_wsBwdData, wsBwdDataSz));

        if (isLern){
            cuCHECK(cudaMalloc(&gpuPrm->d_dw, prms.fWidth * prms.fHeight * insz.d * outsz.d * sizeof(snFloat)));
            cuCHECK(cudaMalloc(&gpuPrm->d_grout, insz.size() * sizeof(snFloat)));
            cuCHECK(cudaMalloc(&gpuPrm->d_wsFwd, wsFwdSz));
            cuCHECK(cudaMalloc(&gpuPrm->d_wsBwdW, wsBwdWSz));
            cuCHECK(cudaMalloc(&gpuPrm->d_bias, insz.d * sizeof(snFloat)));
        }
    }
    else if (!gpuClearMem_ && (gpuPrm->inszMem < insz.size())){
        cuCHECK(cudaFree(gpuPrm->d_in));        gpuPrm->d_in = 0;
        cuCHECK(cudaFree(gpuPrm->d_w));         gpuPrm->d_w = 0;
        cuCHECK(cudaFree(gpuPrm->d_out));       gpuPrm->d_out = 0;
        cuCHECK(cudaFree(gpuPrm->d_wsBwdData)); gpuPrm->d_wsBwdData = 0;

        cuCHECK(cudaMalloc(&gpuPrm->d_in, insz.size() * sizeof(snFloat)));
        cuCHECK(cudaMalloc(&gpuPrm->d_w, prms.fWidth * prms.fHeight * insz.d * outsz.d * sizeof(snFloat)));
        cuCHECK(cudaMalloc(&gpuPrm->d_out, outsz.size() * sizeof(snFloat)));
        cuCHECK(cudaMalloc(&gpuPrm->d_wsBwdData, wsBwdDataSz));

        if (isLern){
            cuCHECK(cudaFree(gpuPrm->d_dw));        gpuPrm->d_dw = 0;
            cuCHECK(cudaFree(gpuPrm->d_grout));     gpuPrm->d_grout = 0;
            cuCHECK(cudaFree(gpuPrm->d_wsFwd));     gpuPrm->d_wsFwd = 0;
            cuCHECK(cudaFree(gpuPrm->d_wsBwdW));    gpuPrm->d_wsBwdW = 0;
            cuCHECK(cudaFree(gpuPrm->d_bias));      gpuPrm->d_bias = 0;

            cuCHECK(cudaMalloc(&gpuPrm->d_dw, prms.fWidth * prms.fHeight * insz.d * outsz.d * sizeof(snFloat)));
            cuCHECK(cudaMalloc(&gpuPrm->d_grout, insz.size() * sizeof(snFloat)));
            cuCHECK(cudaMalloc(&gpuPrm->d_wsFwd, wsFwdSz));
            cuCHECK(cudaMalloc(&gpuPrm->d_wsBwdW, wsBwdWSz));
            cuCHECK(cudaMalloc(&gpuPrm->d_bias, insz.d * sizeof(snFloat)));
        }
        gpuPrm->inszMem = insz.size();
    }
}

void Deconvolution::freeParamCUDA(void* gpuPrms){

    cudaSetDevice(gpuDeviceId_);

    gpuParams* gpuPrm = (gpuParams*)gpuPrms;

    if (!gpuPrm) return;

    cuCHECK(cudnnDestroy(gpuPrm->cudnn));
    cuCHECK(cudnnDestroyConvolutionDescriptor(gpuPrm->conv_desc));
    cuCHECK(cudnnDestroyTensorDescriptor(gpuPrm->in_desc));
    cuCHECK(cudnnDestroyTensorDescriptor(gpuPrm->out_desc));   
    cuCHECK(cudnnDestroyFilterDescriptor(gpuPrm->w_desc));
        
    cuCHECK(cudaFree(gpuPrm->d_in));
    cuCHECK(cudaFree(gpuPrm->d_w));
    cuCHECK(cudaFree(gpuPrm->d_out));
    cuCHECK(cudaFree(gpuPrm->d_wsBwdData));

    if (gpuPrm->grin_desc){ // isLern
        cuCHECK(cudnnDestroyTensorDescriptor(gpuPrm->grin_desc));
        cuCHECK(cudnnDestroyTensorDescriptor(gpuPrm->grout_desc));
        cuCHECK(cudnnDestroyFilterDescriptor(gpuPrm->dw_desc));
        cuCHECK(cudnnDestroyTensorDescriptor(gpuPrm->bias_desc));

        cuCHECK(cudaFree(gpuPrm->d_dw));
        cuCHECK(cudaFree(gpuPrm->d_bias));
        cuCHECK(cudaFree(gpuPrm->d_grout));
        cuCHECK(cudaFree(gpuPrm->d_wsFwd));
        cuCHECK(cudaFree(gpuPrm->d_wsBwdW));
    }
}

void Deconvolution::forwardCUDA(const deconvParams& prms,
    snFloat* weight, const snSize& insz, snFloat* input, const snSize& outsz, snFloat* output, void* gpuPrms){

    cudaSetDevice(gpuDeviceId_);

    gpuParams* gpuPrm = (gpuParams*)gpuPrms;
    size_t isz = insz.size(), osz = outsz.size();
    if (gpuClearMem_){
        cuCHECK(cudaMalloc(&gpuPrm->d_in, isz * sizeof(snFloat)));
        cuCHECK(cudaMalloc(&gpuPrm->d_w, prms.fWidth * prms.fHeight * insz.d * outsz.d * sizeof(snFloat)));
        cuCHECK(cudaMalloc(&gpuPrm->d_out, osz * sizeof(snFloat)));
        cuCHECK(cudaMalloc(&gpuPrm->d_wsBwdData, gpuPrm->wsBwdDataSz));
    }

    // input
    cuCHECK(cudaMemcpy(gpuPrm->d_in, input, isz * sizeof(snFloat), cudaMemcpyHostToDevice));

    // weight
    size_t wsz = outsz.d * insz.d * prms.fHeight * prms.fWidth;
    cuCHECK(cudaMemcpy(gpuPrm->d_w, weight, wsz * sizeof(snFloat), cudaMemcpyHostToDevice));
    
    // run
    snFloat alpha = 1.f, beta = 0.f;
    cuCHECK(cudnnConvolutionBackwardData(gpuPrm->cudnn,
        &alpha,
        gpuPrm->w_desc,
        gpuPrm->d_w,
        gpuPrm->in_desc,
        gpuPrm->d_in,
        gpuPrm->conv_desc,
        gpuPrm->algoBwdData,
        gpuPrm->d_wsBwdData,
        gpuPrm->wsBwdDataSz,
        &beta,
        gpuPrm->out_desc,
        gpuPrm->d_out));
        
    // result
    cuCHECK(cudaMemcpy(output, gpuPrm->d_out, osz * sizeof(snFloat), cudaMemcpyDeviceToHost));

    if (gpuClearMem_){
        cuCHECK(cudaFree(gpuPrm->d_in));        gpuPrm->d_in = 0;
        cuCHECK(cudaFree(gpuPrm->d_w));         gpuPrm->d_w = 0;
        cuCHECK(cudaFree(gpuPrm->d_wsBwdData)); gpuPrm->d_wsBwdData = 0;
        cuCHECK(cudaFree(gpuPrm->d_out));       gpuPrm->d_out = 0;
    }
}

__global__ void cuBwdBias(snSize insz, snFloat* bias, snFloat* grout){

    size_t isz = insz.w * insz.h;

    snFloat* pGrOut = grout + isz * insz.d * blockIdx.x;
    unsigned int d = threadIdx.x;
    while (d < insz.d){

        snFloat b = bias[d];
        for (size_t j = 0; j < isz; ++j)
            pGrOut[j] += b;

        pGrOut += isz * blockDim.x;

        d += blockDim.x;
    }
}

void Deconvolution::backwardCUDA_GW(const deconvParams& prms,
    snFloat* weight, const snSize& insz, snFloat* input, const snSize& outsz, snFloat* gradIn, snFloat* gradOut, snFloat* dWeightOut, void* gpuPrms){

    cudaSetDevice(gpuDeviceId_);

    gpuParams* gpuPrm = (gpuParams*)gpuPrms;
    size_t isz = insz.size(), osz = outsz.size();
    void* d_grin = gpuPrm->d_out;
    if (gpuClearMem_){
        cuCHECK(cudaMalloc(&gpuPrm->d_in, isz * sizeof(snFloat)));
        cuCHECK(cudaMalloc(&gpuPrm->d_w, prms.fWidth * prms.fHeight * insz.d * outsz.d * sizeof(snFloat)));
        cuCHECK(cudaMalloc(&gpuPrm->d_dw, prms.fWidth * prms.fHeight * insz.d * outsz.d * sizeof(snFloat)));
        cuCHECK(cudaMalloc(&gpuPrm->d_bias, insz.d * sizeof(snFloat)));
        cuCHECK(cudaMalloc(&d_grin, osz * sizeof(snFloat)));
        cuCHECK(cudaMalloc(&gpuPrm->d_grout, isz * sizeof(snFloat)));
        cuCHECK(cudaMalloc(&gpuPrm->d_wsFwd, gpuPrm->wsFwdSz));
        cuCHECK(cudaMalloc(&gpuPrm->d_wsBwdW, gpuPrm->wsBwdWSz));
    }

    // input
    cuCHECK(cudaMemcpy(gpuPrm->d_in, input, isz * sizeof(snFloat), cudaMemcpyHostToDevice));

    // grin
    cuCHECK(cudaMemcpy(d_grin, gradIn, osz * sizeof(snFloat), cudaMemcpyHostToDevice));

    // weight
    size_t wsz = outsz.d * insz.d * prms.fHeight * prms.fWidth;
    cuCHECK(cudaMemcpy(gpuPrm->d_w, weight, wsz * sizeof(snFloat), cudaMemcpyHostToDevice));
  
    // run       
    snFloat alpha = 1.f, beta = 0.f;
    cuCHECK(cudnnConvolutionForward(gpuPrm->cudnn,
        &alpha,
        gpuPrm->grin_desc,
        d_grin,
        gpuPrm->w_desc,
        gpuPrm->d_w,
        gpuPrm->conv_desc,
        gpuPrm->algoFwd,
        gpuPrm->d_wsFwd,
        gpuPrm->wsFwdSz,
        &beta,
        gpuPrm->grout_desc,
        gpuPrm->d_grout));

    cuCHECK(cudnnConvolutionBackwardFilter(gpuPrm->cudnn,
        &alpha,        
        gpuPrm->grin_desc,
        d_grin,
        gpuPrm->in_desc,
        gpuPrm->d_in,
        gpuPrm->conv_desc,
        gpuPrm->algoBwdW,
        gpuPrm->d_wsBwdW,
        gpuPrm->wsBwdWSz,
        &beta,
        gpuPrm->dw_desc,
        gpuPrm->d_dw));

    cuCHECK(cudnnConvolutionBackwardBias(gpuPrm->cudnn,
        &alpha,
        gpuPrm->in_desc,
        gpuPrm->d_in,
        &beta,
        gpuPrm->bias_desc,
        gpuPrm->d_bias));

    // +bias
    cuBwdBias <<< int(insz.n), 128 >>> (insz, gpuPrm->d_bias, gpuPrm->d_grout);

    // result
    cuCHECK(cudaMemcpy(gradOut, gpuPrm->d_grout, isz * sizeof(snFloat), cudaMemcpyDeviceToHost));
    cuCHECK(cudaMemcpy(dWeightOut, gpuPrm->d_dw, wsz * sizeof(snFloat), cudaMemcpyDeviceToHost));
    cuCHECK(cudaMemcpy(dWeightOut + wsz, gpuPrm->d_bias, insz.d * sizeof(snFloat), cudaMemcpyDeviceToHost));

    if (gpuClearMem_){
        cuCHECK(cudaFree(gpuPrm->d_in));         gpuPrm->d_in = 0;
        cuCHECK(cudaFree(gpuPrm->d_w));          gpuPrm->d_w = 0;
        cuCHECK(cudaFree(d_grin));               gpuPrm->d_out = 0;
        cuCHECK(cudaFree(gpuPrm->d_grout));      gpuPrm->d_grout = 0;
        cuCHECK(cudaFree(gpuPrm->d_dw));         gpuPrm->d_dw = 0;
        cuCHECK(cudaFree(gpuPrm->d_bias));       gpuPrm->d_bias = 0;
        cuCHECK(cudaFree(gpuPrm->d_wsFwd));      gpuPrm->d_wsFwd = 0;
        cuCHECK(cudaFree(gpuPrm->d_wsBwdW));     gpuPrm->d_wsBwdW = 0;
    }
}

void Deconvolution::backwardCUDA_G(const deconvParams& prms,
    snFloat* weight, const snSize& insz, const snSize& outsz, snFloat* gradIn, snFloat* gradOut, void* gpuPrms){

    cudaSetDevice(gpuDeviceId_);


    gpuParams* gpuPrm = (gpuParams*)gpuPrms;
    size_t isz = insz.size(), osz = outsz.size();
    void* d_grin = gpuPrm->d_out;
    if (gpuClearMem_){
        cuCHECK(cudaMalloc(&gpuPrm->d_w, prms.fWidth * prms.fHeight * insz.d * outsz.d * sizeof(snFloat)));
        cuCHECK(cudaMalloc(&d_grin, osz * sizeof(snFloat)));
        cuCHECK(cudaMalloc(&gpuPrm->d_grout, isz * sizeof(snFloat)));
        cuCHECK(cudaMalloc(&gpuPrm->d_wsFwd, gpuPrm->wsFwdSz));
        cuCHECK(cudaMalloc(&gpuPrm->d_bias, insz.d * sizeof(snFloat)));
    }

    // grin
    cuCHECK(cudaMemcpy(d_grin, gradIn, osz * sizeof(snFloat), cudaMemcpyHostToDevice));

    // weight
    size_t wsz = outsz.d * insz.d * prms.fHeight * prms.fWidth;
    cuCHECK(cudaMemcpy(gpuPrm->d_w, weight, wsz * sizeof(snFloat), cudaMemcpyHostToDevice));
    cuCHECK(cudaMemcpy(gpuPrm->d_bias, weight + wsz, insz.d * sizeof(snFloat), cudaMemcpyHostToDevice));

    // run      
    snFloat alpha = 1.f, beta = 0.f;
    cuCHECK(cudnnConvolutionForward(gpuPrm->cudnn,
        &alpha,
        gpuPrm->grin_desc,
        d_grin,
        gpuPrm->w_desc,
        gpuPrm->d_w,
        gpuPrm->conv_desc,
        gpuPrm->algoFwd,
        gpuPrm->d_wsFwd,
        gpuPrm->wsFwdSz,
        &beta,
        gpuPrm->grout_desc,
        gpuPrm->d_grout));

    // +bias
    cuBwdBias <<< int(insz.n), 128 >>> (insz, gpuPrm->d_bias, gpuPrm->d_grout);

    // результ
    cuCHECK(cudaMemcpy(gradOut, gpuPrm->d_grout, isz * sizeof(snFloat), cudaMemcpyDeviceToHost));

    if (gpuClearMem_){      
        cuCHECK(cudaFree(gpuPrm->d_w));          gpuPrm->d_w = 0;
        cuCHECK(cudaFree(d_grin));               gpuPrm->d_out = 0;
        cuCHECK(cudaFree(gpuPrm->d_bias));       gpuPrm->d_bias = 0;
        cuCHECK(cudaFree(gpuPrm->d_grout));      gpuPrm->d_grout = 0;
        cuCHECK(cudaFree(gpuPrm->d_wsFwd));      gpuPrm->d_wsFwd = 0;
    }
}


#elif SN_CUDA

#include <cuda_runtime.h>
#include "../stdafx.h"
#include "snOperator/src/Operator/deconvolution.h"

using namespace std;
using namespace SN_Base;

#ifndef cuCHECK
#define cuCHECK(func) if (func != 0){ ERROR_MESS("CUDA error: " + cudaGetErrorString(cudaGetLastError())); return;}
#endif

struct gpuParams{
    
    snFloat* d_in = 0;
    snFloat* d_w = 0;
    snFloat* d_dw = 0;
    snFloat* d_out = 0;
    snFloat* d_grout = 0;

    size_t inszMem = 0;
};

void Deconvolution::iniParamCUDA(bool isLern, const snSize& insz, const snSize& outsz, const deconvParams& prms, void** pGpuPrm){
   
    cudaSetDevice(gpuDeviceId_);

    gpuParams* gpuPrm = (gpuParams*)*pGpuPrm;
    if (!gpuPrm){

        cudaDeviceProp cu_deviceProps;
        cudaGetDeviceProperties(&cu_deviceProps, 0);
        if (cu_deviceProps.major < 3){
            ERROR_MESS("%s requires SM >= 3.0");
            return;
        }
        gpuPrm = new gpuParams();
        memset(gpuPrm, 0, sizeof(gpuParams));
        *pGpuPrm = gpuPrm;
               
        if (!gpuClearMem_){
            cuCHECK(cudaMalloc(&gpuPrm->d_in, insz.size() * sizeof(snFloat)));
            cuCHECK(cudaMalloc(&gpuPrm->d_w, (prms.fWidth * prms.fHeight * outsz.d + 1) * insz.d * sizeof(snFloat)));
            cuCHECK(cudaMalloc(&gpuPrm->d_out, outsz.size() * sizeof(snFloat)));

            if (isLern){
                cuCHECK(cudaMalloc(&gpuPrm->d_grout, insz.size() * sizeof(snFloat)));
                cuCHECK(cudaMalloc(&gpuPrm->d_dw, (prms.fWidth * prms.fHeight * outsz.d + 1) * insz.d * outsz.n * sizeof(snFloat)));
            }
        }
    }
    else if (!gpuClearMem_ && (gpuPrm->inszMem < insz.size())){
       
        cuCHECK(cudaFree(gpuPrm->d_in));    gpuPrm->d_in = 0;
        cuCHECK(cudaFree(gpuPrm->d_w));     gpuPrm->d_w = 0;
        cuCHECK(cudaFree(gpuPrm->d_out));   gpuPrm->d_out = 0;

        cuCHECK(cudaMalloc(&gpuPrm->d_in, insz.size() * sizeof(snFloat)));
        cuCHECK(cudaMalloc(&gpuPrm->d_w, (prms.fWidth * prms.fHeight * outsz.d + 1) * insz.d * sizeof(snFloat)));
        cuCHECK(cudaMalloc(&gpuPrm->d_out, outsz.size() * sizeof(snFloat)));
       
        if (isLern){
            cuCHECK(cudaFree(gpuPrm->d_grout)); gpuPrm->d_grout = 0;
            cuCHECK(cudaFree(gpuPrm->d_dw));    gpuPrm->d_dw = 0;

            cuCHECK(cudaMalloc(&gpuPrm->d_grout, insz.size() * sizeof(snFloat)));
            cuCHECK(cudaMalloc(&gpuPrm->d_dw, (prms.fWidth * prms.fHeight * outsz.d + 1) * insz.d * outsz.n * sizeof(snFloat)));
        }
        gpuPrm->inszMem = insz.size();
    }
}

void Deconvolution::freeParamCUDA(void* gpuPrms){
   
    cudaSetDevice(gpuDeviceId_);

    gpuParams* gpuPrm = (gpuParams*)gpuPrms;

    if (!gpuPrm) return;
       
    cuCHECK(cudaFree(gpuPrm->d_in));
    cuCHECK(cudaFree(gpuPrm->d_w));
    cuCHECK(cudaFree(gpuPrm->d_out));
    
    if (gpuPrm->d_grout){ // isLern
        cuCHECK(cudaFree(gpuPrm->d_grout));
        cuCHECK(cudaFree(gpuPrm->d_dw));
    }
}

__global__ void cuDeconvFwd(size_t fWidth, size_t fHeight, size_t stride,
    snFloat* weight, snSize insz, snFloat* input, snSize outsz, snFloat* output){

    size_t wStepByD = fWidth * fHeight,       
        wStepByK = wStepByD * outsz.d,    
        outStepByD = outsz.w * outsz.h,       
        outStepByN = outStepByD * outsz.d,    
        inStepByD = insz.w * insz.h,          
        inStepByN = inStepByD * insz.d;       

    // gridDim.x - number of output layers
    // gridDim.y - batch sz

    weight += blockIdx.x * wStepByD;
    input += blockIdx.y * inStepByN;
    output += blockIdx.x * outStepByD + blockIdx.y * outStepByN;


    unsigned int oz = 0;
    while (oz < insz.d){

        unsigned int oy = threadIdx.y;
        while (oy < insz.h){

            unsigned int ox = threadIdx.x;
            while (ox < insz.w){

                size_t posW = ox * stride, posH = oy * stride;
                               
                // kernel
                snFloat in = input[ox + oy * insz.w];
#pragma unroll
                for (size_t c = 0; c < wStepByD; ++c){

                    size_t cx = c % fWidth, cy = c / fWidth,
                        si = (cx + posW) + (cy + posH) * outsz.w,
                        sw = cx + cy * fWidth;

                    output[si] += in * weight[sw];
                }

                ox += blockDim.x;
            }
            oy += blockDim.y;
        }
        weight += wStepByK;
        input += inStepByD;
        ++oz;
    }
}

void Deconvolution::forwardCUDA(const deconvParams& prms,
    snFloat* weight, const snSize& insz, snFloat* input, const snSize& outsz, snFloat* output, void* gpuPrms){
    
    cudaSetDevice(gpuDeviceId_);
          
    gpuParams* gpuPrm = (gpuParams*)gpuPrms;
    size_t isz = insz.size(), osz = outsz.size(),
           wsz = (prms.fWidth * prms.fHeight * outsz.d + 1) * insz.d;
    if (gpuClearMem_){
        cuCHECK(cudaMalloc(&gpuPrm->d_in, isz * sizeof(snFloat)));
        cuCHECK(cudaMalloc(&gpuPrm->d_w, wsz * sizeof(snFloat)));
        cuCHECK(cudaMalloc(&gpuPrm->d_out, osz * sizeof(snFloat)));
    }

    // input
    cuCHECK(cudaMemcpy(gpuPrm->d_in, input, isz * sizeof(snFloat), cudaMemcpyHostToDevice));

    // weight
    cuCHECK(cudaMemcpy(gpuPrm->d_w, weight, wsz * sizeof(snFloat), cudaMemcpyHostToDevice));

    // out
    cuCHECK(cudaMemset(gpuPrm->d_out, 0, osz * sizeof(snFloat)));

    // run     
    dim3 dimBlock(16, 16);
    dim3 dimGrid(int(outsz.d), int(outsz.n));

    cuDeconvFwd <<< dimGrid, dimBlock >>>(prms.fWidth, 
        prms.fHeight,
        prms.stride,
        gpuPrm->d_w,
        insz, 
        gpuPrm->d_in,
        outsz, 
        gpuPrm->d_out);

    // result
    cuCHECK(cudaMemcpy(output, gpuPrm->d_out, osz * sizeof(snFloat), cudaMemcpyDeviceToHost));

    if (gpuClearMem_){
        cuCHECK(cudaFree(gpuPrm->d_in));   gpuPrm->d_in = 0;
        cuCHECK(cudaFree(gpuPrm->d_w));    gpuPrm->d_w = 0;
        cuCHECK(cudaFree(gpuPrm->d_out));  gpuPrm->d_out = 0;
    }
}

__global__ void cuDeconvBwd_GW(size_t fWidth, size_t fHeight, size_t stride,
    snFloat* weight, snSize insz, snFloat* input, snSize outsz, snFloat* gradIn, snFloat* gradOut, snFloat* dWeightOut){

    size_t wStepByD = fWidth * fHeight,      
        wStepByK = wStepByD * outsz.d,   
        wStepByN = wStepByK * insz.d + insz.d,
        outStepByD = outsz.w * outsz.h,      
        outStepByN = outStepByD * outsz.d,   
        inStepByD = insz.w * insz.h,         
        inStepByN = inStepByD * insz.d;      

    // gridDim.x - number of input layers
    // gridDim.y - batch sz

    input += blockIdx.x * inStepByD + blockIdx.y * inStepByN;
    weight += blockIdx.x * wStepByK;
    dWeightOut += blockIdx.x * wStepByK + blockIdx.y * wStepByN;
    gradIn += blockIdx.y * outStepByN;
    gradOut += blockIdx.x * inStepByD + blockIdx.y * inStepByN;

    unsigned int oz = 0;
    while (oz < outsz.d){
             
        unsigned int oy = threadIdx.y;
        while (oy < insz.h){

            unsigned int ox = threadIdx.x;
            while (ox < insz.w){

                if (oz == 0)
                    gradOut[ox + oy * insz.w] = weight[wStepByK * (insz.d - blockIdx.x) + blockIdx.x]; // bias

                size_t posW = ox * stride, posH = oy * stride;

                // kernel 
                snFloat csum = 0, cin = input[ox + oy * insz.w];
#pragma unroll
                for (size_t c = 0; c < wStepByD; ++c){

                    size_t cx = c % fWidth, cy = c / fWidth,
                        si = (cx + posW) + (cy + posH) * outsz.w,
                        sw = cx + cy * fWidth;

                    csum += gradIn[si] * weight[sw];

                    dWeightOut[sw] += gradIn[si] * cin;
                }
                gradOut[ox + oy * insz.w] += csum;

                if (oz == 0)
                    dWeightOut[wStepByK * (insz.d - blockIdx.x) + blockIdx.x] += cin; // bias

                ox += blockDim.x;
            }
            oy += blockDim.y;
        }
        weight += wStepByD;
        dWeightOut += wStepByD;
        gradIn += outStepByD;
        ++oz;
    }
}

__global__ void cuDeconvWeightMean(size_t kernel, size_t fWidth, size_t fHeight, snSize insz, snFloat* weight){

    size_t wStepByD = fWidth * fHeight,     
        wStepByK = wStepByD * kernel,   
        wStepByN = wStepByK * insz.d + insz.d;
        
    unsigned int ox = threadIdx.x;
    while (ox < wStepByN){

        snFloat csum = weight[ox];
        for (size_t i = 1; i < insz.n; ++i)
            csum += weight[ox + wStepByN * i];
               
        weight[ox] = csum / insz.n;

        ox += blockDim.x;
    }   
}

void Deconvolution::backwardCUDA_GW(const deconvParams& prms,
    snFloat* weight, const snSize& insz, snFloat* input, const snSize& outsz, snFloat* gradIn, snFloat* gradOut, snFloat* dWeightOut, void* gpuPrms){
    
    cudaSetDevice(gpuDeviceId_);

    gpuParams* gpuPrm = (gpuParams*)gpuPrms;
    size_t isz = insz.size(), osz = outsz.size(), 
           wsz = (prms.fWidth * prms.fHeight * outsz.d + 1) * insz.d;
    snFloat* d_grin = gpuPrm->d_out;
    if (gpuClearMem_){
        cuCHECK(cudaMalloc(&gpuPrm->d_in, isz * sizeof(snFloat)));                                         
        cuCHECK(cudaMalloc(&gpuPrm->d_w, wsz * sizeof(snFloat)));
        cuCHECK(cudaMalloc(&d_grin, osz * sizeof(snFloat)));
        cuCHECK(cudaMalloc(&gpuPrm->d_grout, isz * sizeof(snFloat)));                                      
        cuCHECK(cudaMalloc(&gpuPrm->d_dw, wsz * outsz.n * sizeof(snFloat)));
    }

    // input
    cuCHECK(cudaMemcpy(gpuPrm->d_in, input, isz * sizeof(snFloat), cudaMemcpyHostToDevice));

    cuCHECK(cudaMemcpy(d_grin, gradIn, osz * sizeof(snFloat), cudaMemcpyHostToDevice));

    // weight
    cuCHECK(cudaMemcpy(gpuPrm->d_w, weight, wsz * sizeof(snFloat), cudaMemcpyHostToDevice));
     
    // out 
    cuCHECK(cudaMemset(gpuPrm->d_dw, 0, wsz * outsz.n * sizeof(snFloat)));

    // run   
    dim3 dimBlock(16, 16);
    dim3 dimGrid(int(insz.d), int(outsz.n));
   
    cuDeconvBwd_GW <<< dimGrid, dimBlock >>> (prms.fWidth,
        prms.fHeight, 
        prms.stride,
        gpuPrm->d_w,
        insz,
        gpuPrm->d_in,
        outsz, d_grin, 
        gpuPrm->d_grout, 
        gpuPrm->d_dw);

    cuDeconvWeightMean <<< 1, 32 >>> (prms.kernel, prms.fWidth, prms.fHeight, insz, gpuPrm->d_dw);
   
    // result
    cuCHECK(cudaMemcpy(gradOut, gpuPrm->d_grout, isz * sizeof(snFloat), cudaMemcpyDeviceToHost));
    cuCHECK(cudaMemcpy(dWeightOut, gpuPrm->d_dw, wsz * sizeof(snFloat), cudaMemcpyDeviceToHost));

    if (gpuClearMem_){
        cuCHECK(cudaFree(gpuPrm->d_in));      gpuPrm->d_in = 0;
        cuCHECK(cudaFree(gpuPrm->d_w));       gpuPrm->d_w = 0;
        cuCHECK(cudaFree(d_grin));            gpuPrm->d_out = 0;
        cuCHECK(cudaFree(gpuPrm->d_grout));   gpuPrm->d_grout = 0;
        cuCHECK(cudaFree(gpuPrm->d_dw));      gpuPrm->d_dw = 0;
    }
}

__global__ void cuDeconvBwd_G(size_t fWidth, size_t fHeight, size_t stride,
    snFloat* weight, snSize insz, snSize outsz, snFloat* gradIn, snFloat* gradOut){

    size_t wStepByD = fWidth * fHeight,     
        wStepByK = wStepByD * outsz.d,  
        outStepByD = outsz.w * outsz.h,     
        outStepByN = outStepByD * outsz.d,  
        inStepByD = insz.w * insz.h,        
        inStepByN = inStepByD * insz.d;     

    // gridDim.x - number of input layers
    // gridDim.y - batch size

    weight += blockIdx.x * wStepByK;
    gradIn += blockIdx.y * outStepByN;
    gradOut += blockIdx.x * inStepByD + blockIdx.y * inStepByN;

    unsigned int oz = 0;
    while (oz < outsz.d){

        unsigned int oy = threadIdx.y;
        while (oy < insz.h){

            unsigned int ox = threadIdx.x;
            while (ox < insz.w){

                if (oz == 0)
                    gradOut[ox + oy * insz.w] = weight[wStepByK * (insz.d - blockIdx.x) + blockIdx.x]; // bias

                size_t posW = ox * stride, posH = oy * stride;

                // kernel
                snFloat csum = 0;
#pragma unroll
                for (size_t c = 0; c < wStepByD; ++c){

                    size_t cx = c % fWidth, cy = c / fWidth,
                        si = (cx + posW) + (cy + posH) * outsz.w,
                        sw = cx + cy * fWidth;

                    csum += gradIn[si] * weight[sw];
                }
                gradOut[ox + oy * insz.w] += csum;

                ox += blockDim.x;
            }
            oy += blockDim.y;
        }
        weight += wStepByD;
        gradIn += outStepByD;
        ++oz;
    }
}

void Deconvolution::backwardCUDA_G(const deconvParams& prms,
    snFloat* weight, const snSize& insz, const snSize& outsz, snFloat* gradIn, snFloat* gradOut, void* gpuPrms){
    
    cudaSetDevice(gpuDeviceId_);

    gpuParams* gpuPrm = (gpuParams*)gpuPrms;
    size_t isz = insz.size(), osz = outsz.size(), 
           wsz = (prms.fWidth * prms.fHeight * outsz.d + 1) * insz.d;
    snFloat* d_grin = gpuPrm->d_out;
    if (gpuClearMem_){
        cuCHECK(cudaMalloc(&d_grin, osz * sizeof(snFloat)));
        cuCHECK(cudaMalloc(&gpuPrm->d_w, wsz * sizeof(snFloat)));
        cuCHECK(cudaMalloc(&gpuPrm->d_grout, isz * sizeof(snFloat)));
    }

    // input
    cuCHECK(cudaMemcpy(d_grin, gradIn, osz * sizeof(snFloat), cudaMemcpyHostToDevice));

    // weight
    cuCHECK(cudaMemcpy(gpuPrm->d_w, weight, wsz * sizeof(snFloat), cudaMemcpyHostToDevice));
        
    
    // run   
    dim3 dimBlock(16, 16);
    dim3 dimGrid(int(insz.d), int(outsz.n));

    cuDeconvBwd_G <<< dimGrid, dimBlock >>> (prms.fWidth,
        prms.fHeight,
        prms.stride, 
        gpuPrm->d_w, 
        insz, 
        outsz,
        d_grin, 
        gpuPrm->d_grout);
       
    // result
    cuCHECK(cudaMemcpy(gradOut, gpuPrm->d_grout, isz * sizeof(snFloat), cudaMemcpyDeviceToHost));

    if (gpuClearMem_){
        cuCHECK(cudaFree(d_grin));            gpuPrm->d_out = 0;
        cuCHECK(cudaFree(gpuPrm->d_w));       gpuPrm->d_w = 0;
        cuCHECK(cudaFree(gpuPrm->d_grout));   gpuPrm->d_grout = 0;
    }
}

#endif 
